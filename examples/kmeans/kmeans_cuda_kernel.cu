#include "hip/hip_runtime.h"
#ifndef _KMEANS_CUDA_KERNEL_H_
#define _KMEANS_CUDA_KERNEL_H_

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "kmeans.h"

#ifndef FLT_MAX
#define FLT_MAX 3.40282347e+38f
#endif

// *** texture/constant 메모리 전부 제거 ***

/* ----------------- invert_mapping() --------------------- */
/* Row-major (AoS) → Column-major (SoA) 변환.
   input:  [p0,d0][p0,d1]...[p1,d0][p1,d1]...
   output: [d0,p0][d0,p1]...[d1,p0][d1,p1]...
*/
__global__ void invert_mapping(float *input, float *output,
                               int npoints, int nfeatures)
{
    int point_id = threadIdx.x + blockDim.x * blockIdx.x;
    if (point_id < npoints){
        for (int i=0;i<nfeatures;i++)
            output[point_id + (size_t)npoints * i] =
                input[(size_t)point_id * nfeatures + i];
    }
}

/* ----------------- kmeansPoint() --------------------- */
/* 각 포인트의 최근접 클러스터를 찾아 membership을 갱신
   features: SoA [nfeatures * npoints]
   clusters: Row-major [nclusters * nfeatures]
*/
__global__ void kmeansPoint(float  *features,   // SoA
                            int     nfeatures,
                            int     npoints,
                            int     nclusters,
                            int    *membership,
                            float  *clusters,   // 글로벌 메모리에서 직접 읽음
                            float  *block_clusters, // 그대로 유지(옵션)
                            int    *block_deltas)   // 그대로 유지(옵션)
{
    // block/point id 계산 (Rodinia 원형 유지)
    const unsigned int block_id =
        gridDim.x * blockIdx.y + blockIdx.x;
    const unsigned int point_id =
        block_id * (blockDim.x * blockDim.y) + threadIdx.x;

    int index = -1;

    if (point_id < (unsigned)npoints) {
        float min_dist = FLT_MAX;

        // 모든 클러스터와 거리 비교 (유클리드 제곱거리, sqrt 없음)
        for (int c = 0; c < nclusters; ++c) {
            int cluster_base = c * nfeatures;
            float acc = 0.0f;
            // SoA 인덱싱: features[d * npoints + point_id]
            for (int d = 0; d < nfeatures; ++d) {
                float f = features[(size_t)d * npoints + point_id];
                float diff = f - clusters[cluster_base + d];
                acc += diff * diff;
            }
            if (acc < min_dist) {
                min_dist = acc;
                index = c;
            }
        }
    }

#ifdef GPU_DELTA_REDUCTION
    __shared__ int deltas[THREADS_PER_BLOCK];
    if (threadIdx.x < THREADS_PER_BLOCK) deltas[threadIdx.x] = 0;
#endif

    if (point_id < (unsigned)npoints) {
#ifdef GPU_DELTA_REDUCTION
        if (membership[point_id] != index) deltas[threadIdx.x] = 1;
#endif
        membership[point_id] = index;
    }

#ifdef GPU_DELTA_REDUCTION
    __syncthreads();
    unsigned int live = THREADS_PER_BLOCK / 2;
    for (; live > 1; live >>= 1) {
        if (threadIdx.x < live)
            deltas[threadIdx.x] += deltas[threadIdx.x + live];
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        deltas[0] += deltas[1];
        block_deltas[blockIdx.y * gridDim.x + blockIdx.x] = deltas[0];
    }
#endif

#ifdef GPU_NEW_CENTER_REDUCTION
    // (원형 유지가 필요하면 이 블록 그대로 사용.
    //  여기서는 CPU_CENTER_REDUCE가 기본이라 생략 가능)
    int center_id = threadIdx.x / nfeatures;
    int dim_id    = threadIdx.x - nfeatures * center_id;

    __shared__ int new_center_ids[THREADS_PER_BLOCK];
    new_center_ids[threadIdx.x] = index;
    __syncthreads();

    int new_base = (point_id - threadIdx.x) * nfeatures + dim_id;
    float accumulator = 0.f;

    if (threadIdx.x < nfeatures * nclusters) {
        for (int i = 0; i < THREADS_PER_BLOCK; i++) {
            float val = features[(size_t)dim_id * npoints + (point_id - threadIdx.x + i)];
            if (new_center_ids[i] == center_id) accumulator += val;
        }
        block_clusters[(blockIdx.y * gridDim.x + blockIdx.x)
                       * nclusters * nfeatures + threadIdx.x] = accumulator;
    }
#endif
}
#endif // _KMEANS_CUDA_KERNEL_H_
#include "hip/hip_runtime.h"
// kmeans_cuda.cu
//
// CUDA port that mirrors the common OpenCL k-means benchmark (Rodinia-style).
// It provides drop-in equivalents for the two main kernels:
//  1) find_nearest_cluster: assigns each point to the closest cluster and produces
//     per-block partial sums & counts
//  2) sum_reduce_new_centers: reduces per-block partials into the final new centers
//
// Host helpers are also provided to allocate/launch kernels and to copy results.
//
// Expected memory layout (matches the typical OpenCL version):
// - features: [npoints][nfeatures]   (row-major, contiguous)
// - clusters: [nclusters][nfeatures] (row-major, contiguous)
// - membership: [npoints]
// - partial_new_centers: [nblocks][nclusters][nfeatures]
// - partial_new_centers_len: [nblocks][nclusters]
//
// Build (example):
//   nvcc -O3 -arch=sm_70 -o kmeans_cuda kmeans_cuda.cu
//
// Minimal usage example (host side):
//   See the runKMeansCUDA(...) function below; adapt to your project as needed.
//
// NOTE: If your OpenCL code used local memory tiling sizes or specific NDRange
//       geometry, adjust BLOCK_SIZE and grid sizing to match 1:1 semantics.
//
// (c) CUDA port for parity with OpenCL benchmark.

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <limits>
#include <algorithm>

#ifndef CUDA_CHECK
#define CUDA_CHECK(expr)                                                    \
  do {                                                                      \
    hipError_t _e = (expr);                                                \
    if (_e != hipSuccess) {                                                \
      fprintf(stderr, "CUDA Error %s:%d: %s -> %s\n",                       \
              __FILE__, __LINE__, #expr, hipGetErrorString(_e));           \
      std::abort();                                                         \
    }                                                                       \
  } while (0)
#endif

// Tunable launch parameter (mirrors typical OpenCL local size of 256)
#ifndef BLOCK_SIZE
#define BLOCK_SIZE 256
#endif

// Compute squared distance between two feature vectors of length nfeatures
__device__ inline float distance_sq(const float* __restrict__ p,
                                    const float* __restrict__ c,
                                    int nfeatures) {
  float dist = 0.f;
  for (int f = 0; f < nfeatures; ++f) {
    float d = p[f] - c[f];
    dist += d * d;
  }
  return dist;
}

// Kernel 1: assign each point to nearest cluster and accumulate per-block partial sums
// Layouts:
//   features: [npoints][nfeatures]
//   clusters: [nclusters][nfeatures]
//   membership: [npoints]
//   partial_new_centers: [gridDim.x][nclusters][nfeatures]
//   partial_new_centers_len: [gridDim.x][nclusters]
__global__ void find_nearest_cluster_kernel(const float* __restrict__ features,
                                            const float* __restrict__ clusters,
                                            int npoints,
                                            int nfeatures,
                                            int nclusters,
                                            int* __restrict__ membership,
                                            float* __restrict__ partial_new_centers,
                                            int* __restrict__ partial_new_centers_len) {
  extern __shared__ unsigned char smem[];
  float* s_clusters = reinterpret_cast<float*>(smem); // size = nclusters*nfeatures floats

  // Load clusters into shared memory (all threads cooperate)
  int tid = threadIdx.x;
  int tcount = blockDim.x;
  int total_cf = nclusters * nfeatures;
  for (int i = tid; i < total_cf; i += tcount) {
    s_clusters[i] = clusters[i];
  }
  __syncthreads();

  int global_id = blockIdx.x * blockDim.x + threadIdx.x;
  if (global_id >= npoints) return;

  // Pointer to this point
  const float* p = features + (size_t)global_id * nfeatures;

  // Find nearest cluster
  float min_dist = FLT_MAX;
  int   idx = -1;
  for (int c = 0; c < nclusters; ++c) {
    const float* sc = s_clusters + (size_t)c * nfeatures;
    float d = distance_sq(p, sc, nfeatures);
    if (d < min_dist) {
      min_dist = d;
      idx = c;
    }
  }

  // Update membership
  membership[global_id] = idx;

  // Accumulate into per-block partial sums
  // Compute base pointers for this block
  float* block_partials = partial_new_centers
                        + (size_t)blockIdx.x * nclusters * nfeatures;
  int* block_counts = partial_new_centers_len
                    + (size_t)blockIdx.x * nclusters;

  // Atomically add features into the cluster's partial sum
  float* dst = block_partials + (size_t)idx * nfeatures;
  for (int f = 0; f < nfeatures; ++f) {
    atomicAdd(&dst[f], p[f]);
  }
  // Atomically increment count
  atomicAdd(&block_counts[idx], 1);
}

// Kernel 2: reduce partials across blocks into final new_centers and new_centers_len
// Option A: single-block kernel per cluster (simple + robust)
// new_centers: [nclusters][nfeatures]
// new_centers_len: [nclusters]
__global__ void sum_reduce_new_centers_kernel(const float* __restrict__ partial_new_centers,
                                              const int*   __restrict__ partial_new_centers_len,
                                              int nblocks,
                                              int nclusters,
                                              int nfeatures,
                                              float* __restrict__ new_centers,
                                              int*   __restrict__ new_centers_len) {
  // Each cluster handled by one block (launch with <<<nclusters, BLOCK_SIZE>>>)
  int c = blockIdx.x;
  if (c >= nclusters) return;

  // Reduce counts
  int count_sum = 0;
  for (int b = threadIdx.x; b < nblocks; b += blockDim.x) {
    count_sum += partial_new_centers_len[(size_t)b * nclusters + c];
  }
  __shared__ int s_count;
  // Reduce within block (naive reduction)
  __shared__ int s_counts[BLOCK_SIZE];
  s_counts[threadIdx.x] = count_sum;
  __syncthreads();

  // Parallel reduction for counts
  for (int s = blockDim.x >> 1; s > 0; s >>= 1) {
    if (threadIdx.x < s) {
      s_counts[threadIdx.x] += s_counts[threadIdx.x + s];
    }
    __syncthreads();
  }
  if (threadIdx.x == 0) s_count = s_counts[0];
  __syncthreads();

  if (threadIdx.x == 0) {
    new_centers_len[c] = s_count;
  }

  // Reduce feature sums per feature dimension
  for (int f = 0; f < nfeatures; ++f) {
    float sum = 0.f;
    for (int b = threadIdx.x; b < nblocks; b += blockDim.x) {
      sum += partial_new_centers[((size_t)b * nclusters + c) * nfeatures + f];
    }
    __shared__ float sdata[BLOCK_SIZE];
    sdata[threadIdx.x] = sum;
    __syncthreads();
    for (int s = blockDim.x >> 1; s > 0; s >>= 1) {
      if (threadIdx.x < s) {
        sdata[threadIdx.x] += sdata[threadIdx.x + s];
      }
      __syncthreads();
    }
    if (threadIdx.x == 0) {
      new_centers[(size_t)c * nfeatures + f] = sdata[0];
    }
    __syncthreads();
  }
}

// Host wrapper that mirrors OpenCL flow.
// Inputs:
//   d_features  : device pointer [npoints * nfeatures]
//   d_clusters  : device pointer [nclusters * nfeatures]
// Outputs:
//   d_membership: device pointer [npoints]
//   d_new_centers: device pointer [nclusters * nfeatures]  (sums, not divided)
//   d_new_centers_len: device pointer [nclusters]
//
// The function allocates/clears per-block partials internally (or you can pass your own).
extern "C" void kmeansCUDAAssignAndReduce(const float* d_features,
                                          const float* d_clusters,
                                          int npoints,
                                          int nfeatures,
                                          int nclusters,
                                          int* d_membership,
                                          float* d_new_centers,
                                          int* d_new_centers_len) {
  // Grid sizing similar to typical OpenCL NDRange (global = ceil(npoints/BLOCK_SIZE)*BLOCK_SIZE)
  int blocks = (npoints + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim3 grid(blocks);
  dim3 block(BLOCK_SIZE);

  // Allocate per-block partials
  float* d_partial_new_centers = nullptr;
  int*   d_partial_new_centers_len = nullptr;
  size_t partial_centers_bytes = (size_t)blocks * nclusters * nfeatures * sizeof(float);
  size_t partial_counts_bytes  = (size_t)blocks * nclusters * sizeof(int);

  CUDA_CHECK(hipMalloc(&d_partial_new_centers, partial_centers_bytes));
  CUDA_CHECK(hipMalloc(&d_partial_new_centers_len, partial_counts_bytes));

  CUDA_CHECK(hipMemset(d_partial_new_centers, 0, partial_centers_bytes));
  CUDA_CHECK(hipMemset(d_partial_new_centers_len, 0, partial_counts_bytes));
  CUDA_CHECK(hipMemset(d_new_centers, 0, (size_t)nclusters * nfeatures * sizeof(float)));
  CUDA_CHECK(hipMemset(d_new_centers_len, 0, (size_t)nclusters * sizeof(int)));

  // Shared memory size for clusters
  size_t smem_bytes = (size_t)nclusters * nfeatures * sizeof(float);

  // Kernel 1: assign + partial accumulate
  find_nearest_cluster_kernel<<<grid, block, smem_bytes>>>(
      d_features, d_clusters, npoints, nfeatures, nclusters,
      d_membership, d_partial_new_centers, d_partial_new_centers_len);
  CUDA_CHECK(hipGetLastError());

  // Kernel 2: reduce partials (launch one block per cluster)
  dim3 grid2(nclusters);
  dim3 block2(BLOCK_SIZE);
  sum_reduce_new_centers_kernel<<<grid2, block2>>>(
      d_partial_new_centers, d_partial_new_centers_len,
      blocks, nclusters, nfeatures,
      d_new_centers, d_new_centers_len);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  CUDA_CHECK(hipFree(d_partial_new_centers));
  CUDA_CHECK(hipFree(d_partial_new_centers_len));
}

// Convenience CPU helper to finalize new cluster centers by dividing sums by counts.
// This mirrors the OpenCL host step that normalizes sums into means.
extern "C" void kmeansFinalizeCenters(float* h_new_centers, int* h_new_centers_len,
                                      int nclusters, int nfeatures) {
  for (int c = 0; c < nclusters; ++c) {
    int len = std::max(h_new_centers_len[c], 1); // guard against zero
    float inv = 1.0f / float(len);
    for (int f = 0; f < nfeatures; ++f) {
      h_new_centers[(size_t)c * nfeatures + f] *= inv;
    }
  }
}

// A simple end-to-end demo main (optional).
// Define BUILD_STANDALONE=1 to compile a small runnable example that
// creates random data and runs one assignment+reduce iteration.
#ifdef BUILD_STANDALONE
#include <vector>
int main() {
  const int npoints = 1<<14;     // 16K
  const int nfeatures = 32;
  const int nclusters = 16;

  std::vector<float> h_features((size_t)npoints * nfeatures);
  std::vector<float> h_clusters((size_t)nclusters * nfeatures);
  std::vector<int>   h_membership(npoints, -1);
  std::vector<float> h_new_centers((size_t)nclusters * nfeatures);
  std::vector<int>   h_new_centers_len(nclusters);

  // init random
  for (auto& v : h_features) v = (float)rand() / RAND_MAX;
  for (auto& v : h_clusters) v = (float)rand() / RAND_MAX;

  float *d_features=nullptr, *d_clusters=nullptr, *d_new_centers=nullptr;
  int *d_membership=nullptr, *d_new_centers_len=nullptr;
  CUDA_CHECK(hipMalloc(&d_features, h_features.size()*sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_clusters, h_clusters.size()*sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_membership, h_membership.size()*sizeof(int)));
  CUDA_CHECK(hipMalloc(&d_new_centers, h_new_centers.size()*sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_new_centers_len, h_new_centers_len.size()*sizeof(int)));

  CUDA_CHECK(hipMemcpy(d_features, h_features.data(), h_features.size()*sizeof(float), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_clusters, h_clusters.data(), h_clusters.size()*sizeof(float), hipMemcpyHostToDevice));

  kmeansCUDAAssignAndReduce(d_features, d_clusters, npoints, nfeatures, nclusters,
                            d_membership, d_new_centers, d_new_centers_len);

  CUDA_CHECK(hipMemcpy(h_new_centers.data(), d_new_centers, h_new_centers.size()*sizeof(float), hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(h_new_centers_len.data(), d_new_centers_len, h_new_centers_len.size()*sizeof(int), hipMemcpyDeviceToHost));

  // finalize (divide sums by counts)
  kmeansFinalizeCenters(h_new_centers.data(), h_new_centers_len.data(), nclusters, nfeatures);

  printf("Done. Example first center dims: ");
  for (int f = 0; f < min(5, nfeatures); ++f) {
    printf("%g ", h_new_centers[f]);
  }
  printf("\n");

  hipFree(d_features); hipFree(d_clusters); hipFree(d_membership);
  hipFree(d_new_centers); hipFree(d_new_centers_len);
  return 0;
}
#endif

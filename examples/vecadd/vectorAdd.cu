/* nvcc -O2 -std=c++11 vecadd.cu -o vecadd */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <unistd.h>   // getopt

__global__ void vectorAdd(const float *A, const float *B, float *C,
                          int numElements) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < numElements) {
    C[i] = A[i] + B[i] + 0.0f;
  }
}

static void usage(const char* prog) {
  printf("Usage: %s [-n numElements] [-b blockSize] [-h]\n"
         "  -n  vector length (default 16)\n"
         "  -b  threads per block (default 4)\n"
         "  -h  show this help\n", prog);
}

int main(int argc, char** argv) {
  // ---- CLI ----
  int numElements = 16;   // default as original
  int threadsPerBlock = 4; // default as original

  int opt;
  while ((opt = getopt(argc, argv, "n:b:h")) != -1) {
    switch (opt) {
      case 'n': numElements = atoi(optarg); break;
      case 'b': threadsPerBlock = atoi(optarg); break;
      case 'h': usage(argv[0]); return 0;
      default : usage(argv[0]); return 1;
    }
  }
  if (numElements <= 0 || threadsPerBlock <= 0) {
    fprintf(stderr, "Error: numElements and blockSize must be > 0\n");
    return 1;
  }

  printf("[Vector addition of %d elements] (blockSize=%d)\n",
         numElements, threadsPerBlock);

  size_t size = (size_t)numElements * sizeof(float);

  // ---- Host alloc ----
  float *h_A = (float*)malloc(size);
  float *h_B = (float*)malloc(size);
  float *h_C = (float*)malloc(size);
  if (!h_A || !h_B || !h_C) {
    fprintf(stderr, "Failed to allocate host vectors!\n");
    return 1;
  }

  // ---- Init host data ----
  for (int i = 0; i < numElements; ++i) {
    h_A[i] = rand() / (float)RAND_MAX;
    h_B[i] = rand() / (float)RAND_MAX;
  }

  // ---- Device alloc ----
  float *d_A = NULL, *d_B = NULL, *d_C = NULL;
  hipError_t err;

  err = hipMalloc((void**)&d_A, size);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector A (%s)\n",
            hipGetErrorString(err));
    return 1;
  }
  err = hipMalloc((void**)&d_B, size);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector B (%s)\n",
            hipGetErrorString(err));
    return 1;
  }
  err = hipMalloc((void**)&d_C, size);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector C (%s)\n",
            hipGetErrorString(err));
    return 1;
  }

  // ---- H2D ----
  printf("Copy input data from host to device\n");
  err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "H2D A failed: %s\n", hipGetErrorString(err));
    return 1;
  }
  err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "H2D B failed: %s\n", hipGetErrorString(err));
    return 1;
  }

  // ---- Launch ----
  int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
  printf("CUDA kernel launch with %d blocks of %d threads\n",
         blocksPerGrid, threadsPerBlock);

  vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
  err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
    return 1;
  }
  err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    fprintf(stderr, "hipDeviceSynchronize failed: %s\n",
            hipGetErrorString(err));
    return 1;
  }

  // ---- D2H ----
  printf("Copy output data from device to host\n");
  err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    fprintf(stderr, "D2H C failed: %s\n", hipGetErrorString(err));
    return 1;
  }

  // ---- Verify ----
  for (int i = 0; i < numElements; ++i) {
    if (fabsf(h_A[i] + h_B[i] - h_C[i]) > 1e-5f) {
      fprintf(stderr, "Result verification failed at element %d!\n", i);
      return 1;
    }
  }
  printf("Test PASSED\n");

  // ---- Cleanup ----
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  free(h_A);
  free(h_B);
  free(h_C);
  printf("Done\n");
  return 0;
}
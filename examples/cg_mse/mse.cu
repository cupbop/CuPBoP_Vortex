#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <stdio.h>
#include <random>
#include <iostream>
#include <cassert>

namespace cg = cooperative_groups;

#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", \
                    __FILE__, __LINE__, hipGetErrorString(error)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)


// https://github.com/clu0/unet.cu/blob/1b59e9c074c3516a3286f2c7d27c9d679dc6d042/dev/mse.cu#L8
__global__ void mse_forward_kernel(
    const float* inp, const float* y, float* loss, int N
) {
    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    __shared__ float shared_sum[32]; // max 32 warps
    int num_warps = blockDim.x / 32;
    int warp_id = threadIdx.x / 32;
    int lane_id = threadIdx.x % 32;
    float thread_sum = 0.0f;
    
    for (int i = threadIdx.x; i < N; i += blockDim.x) {
        float diff = inp[i] - y[i];
        thread_sum += diff * diff;
    }
    
    // warp reduce
    float warp_sum = cg::reduce(warp, thread_sum, cg::plus<float>{});
    shared_sum[warp_id] = warp_sum;
    __syncthreads();
    
    warp_sum = (lane_id < num_warps) ? shared_sum[lane_id] : 0.0f;
    float block_sum = cg::reduce(warp, warp_sum, cg::plus<float>{});
    if (threadIdx.x == 0) {
        loss[0] = block_sum / N;
    }
}

float compute_mse_cpu(const float* inp, const float* y, int N) {
    float sum = 0.0f;
    for (int i = 0; i < N; i++) {
        float diff = inp[i] - y[i];
        sum += diff * diff;
    }
    return sum / N;
}

void generate_random_data(float* data, int size, float min = -1.0f, float max = 1.0f) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(min, max);
    
    for (int i = 0; i < size; i++) {
        data[i] = dis(gen);
    }
}

int main() {
    const int N = 1024 * 1024;  // 1M elements
    const int threads_per_block = 256;
    
    float *h_input = new float[N];
    float *h_target = new float[N];
    float h_loss = 0.0f;
    
    generate_random_data(h_input, N);
    generate_random_data(h_target, N);
    
    float *d_input, *d_target, *d_loss;
    CUDA_CHECK(hipMalloc(&d_input, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_target, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_loss, sizeof(float)));
    
    CUDA_CHECK(hipMemcpy(d_input, h_input, N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_target, h_target, N * sizeof(float), hipMemcpyHostToDevice));
    
    mse_forward_kernel<<<32, threads_per_block>>>(d_input, d_target, d_loss, N);
    CUDA_CHECK(hipGetLastError());
    
    CUDA_CHECK(hipMemcpy(&h_loss, d_loss, sizeof(float), hipMemcpyDeviceToHost));
    
    float cpu_loss = compute_mse_cpu(h_input, h_target, N);
    
    printf("GPU MSE Loss: %f\n", h_loss);
    printf("CPU MSE Loss: %f\n", cpu_loss);
    printf("Relative difference: %f%%\n", 100.0f * fabsf(h_loss - cpu_loss) / cpu_loss);
    
    const float tolerance = 1e-3;
    assert(fabsf(h_loss - cpu_loss) / cpu_loss < tolerance && "GPU and CPU results differ too much!");
    printf("Verification passed!\n");
    
    delete[] h_input;
    delete[] h_target;
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_target));
    CUDA_CHECK(hipFree(d_loss));
    
    return 0;
}
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <stdio.h>
#include <random>
#include <iostream>
#include <cassert>

#define THREADS_PER_BLOCK 256

namespace cg = cooperative_groups;

#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", \
                    __FILE__, __LINE__, hipGetErrorString(error)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)


// https://github.com/clu0/unet.cu/blob/1b59e9c074c3516a3286f2c7d27c9d679dc6d042/dev/mse.cu#L8
__global__ void mse_forward_kernel(
    const float* inp, const float* y, float* loss, int N
) {
    int cg_group_size = 32;

    __shared__ float shared_sum[32]; // max 32 warps

    __shared__ int num_warps_array[THREADS_PER_BLOCK];
    __shared__ int warp_id_array[THREADS_PER_BLOCK];
    __shared__ int lane_id_array[THREADS_PER_BLOCK];
    __shared__ float thread_sum_array[THREADS_PER_BLOCK];
    
    // Parallel region #1
    for (int tid = 0; tid < THREADS_PER_BLOCK; tid++) {
        num_warps_array[tid] = THREADS_PER_BLOCK / 32;
        warp_id_array[tid] = tid / 32;
        lane_id_array[tid] = tid % 32;
        thread_sum_array[tid] = 0.0f;

        for (int i = tid; i < N; i += THREADS_PER_BLOCK) {
            float diff = inp[i] - y[i];
            thread_sum_array[tid] += diff * diff;
        }
    }

    // Parallel region #2 (hierarchical collapsing)
    __shared__ float cg_reduce_temp_array[THREADS_PER_BLOCK / 32];
    for (int i = 0; i < THREADS_PER_BLOCK / cg_group_size; i++) {
        float cg_reduce_temp = 0.0f;
        for (int j = 0; j < cg_group_size; j++) {
            int tid = i * cg_group_size + j;
            cg_reduce_temp += thread_sum_array[tid];
        }
        cg_reduce_temp_array[i] = cg_reduce_temp;
    }

    // Parallel region #3 (hierarchical collapsing)
    __shared__ float warp_sum_array[THREADS_PER_BLOCK];
    for (int i = 0; i < THREADS_PER_BLOCK / cg_group_size; i++) {
        for (int j = 0; j < cg_group_size; j++) {
            int tid = i * cg_group_size + j;
            warp_sum_array[tid] = cg_reduce_temp_array[i];

            shared_sum[warp_id_array[tid]] = warp_sum_array[tid];
        }
    }

    // Parallel region #4 (hierarchical collapsing)
    __shared__ float cg_reduce_temp2_array[THREADS_PER_BLOCK / 32];
    for (int i = 0; i < THREADS_PER_BLOCK / cg_group_size; i++) {
        float cg_reduce_temp2 = 0.0f;
        for (int j = 0; j < cg_group_size; j++) {
            int tid = i * cg_group_size + j;
            warp_sum_array[tid] = (lane_id_array[tid] < num_warps_array[tid]) ? shared_sum[lane_id_array[tid]] : 0.0f;
            cg_reduce_temp2 += warp_sum_array[tid];
        }
        cg_reduce_temp2_array[i] = cg_reduce_temp2;
    }

    // Parallel region #5 (hierarchical collapsing)
    __shared__ float block_sum_array[THREADS_PER_BLOCK];
    for (int i = 0; i < THREADS_PER_BLOCK / cg_group_size; i++) {
        for (int j = 0; j < cg_group_size; j++) {
            int tid = i * cg_group_size + j;
            block_sum_array[tid] = cg_reduce_temp2_array[i];

            if (tid == 0) {
                loss[0] = block_sum_array[tid] / N;
            }
        }
    }
}

float compute_mse_cpu(const float* inp, const float* y, int N) {
    float sum = 0.0f;
    for (int i = 0; i < N; i++) {
        float diff = inp[i] - y[i];
        sum += diff * diff;
    }
    return sum / N;
}

void generate_random_data(float* data, int size, float min = -1.0f, float max = 1.0f) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(min, max);
    
    for (int i = 0; i < size; i++) {
        data[i] = dis(gen);
    }
}

int main() {
    const int N = 1024 * 1024;  // 1M elements
    const int threads_per_block = 256;
    
    float *h_input = new float[N];
    float *h_target = new float[N];
    float h_loss = 0.0f;
    
    generate_random_data(h_input, N);
    generate_random_data(h_target, N);
    
    float *d_input, *d_target, *d_loss;
    CUDA_CHECK(hipMalloc(&d_input, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_target, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_loss, sizeof(float)));
    
    CUDA_CHECK(hipMemcpy(d_input, h_input, N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_target, h_target, N * sizeof(float), hipMemcpyHostToDevice));
    
    mse_forward_kernel<<<32, 1>>>(d_input, d_target, d_loss, N);
    CUDA_CHECK(hipGetLastError());
    
    CUDA_CHECK(hipMemcpy(&h_loss, d_loss, sizeof(float), hipMemcpyDeviceToHost));
    
    float cpu_loss = compute_mse_cpu(h_input, h_target, N);
    
    printf("GPU MSE Loss: %f\n", h_loss);
    printf("CPU MSE Loss: %f\n", cpu_loss);
    printf("Relative difference: %f%%\n", 100.0f * fabsf(h_loss - cpu_loss) / cpu_loss);
    
    const float tolerance = 1e-3;
    assert(fabsf(h_loss - cpu_loss) / cpu_loss < tolerance && "GPU and CPU results differ too much!");
    printf("Verification passed!\n");
    
    delete[] h_input;
    delete[] h_target;
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_target));
    CUDA_CHECK(hipFree(d_loss));
    
    return 0;
}
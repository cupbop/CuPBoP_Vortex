// cuda_conv3_blockcfg.cu
// Build: nvcc -O3 -std=c++17 cuda_conv3_blockcfg.cu -o cuda_conv3
// Run examples:
//   ./cuda_conv3 -n 32                    # block=(1,1) (기본값)
//   ./cuda_conv3 -n 256 -bx 16 -by 16     # block=(16,16)
//   ./cuda_conv3 -n 1024 -bx 32 -by 8     # block=(32,8)

#include <cstdio>
#include <cstdlib>
#include <cstdint>
#include <cassert>
#include <cstring>
#include <cmath>
#include <chrono>
#include <vector>
#include <unistd.h>   // getopt
#include <hip/hip_runtime.h>

#define FLOAT_ULP 6
#define KERNEL_NAME "conv3"

static void cleanup();

// CUDA error handling
#define CUDA_CHECK(_expr)                                                         \
  do {                                                                            \
    hipError_t _err = (_expr);                                                   \
    if (_err == hipSuccess) break;                                               \
    fprintf(stderr, "CUDA Error: '%s' returned %d (%s)!\n",                       \
            #_expr, (int)_err, hipGetErrorString(_err));                         \
    cleanup();                                                                    \
    std::exit(-1);                                                                \
  } while (0)

static bool compare_equal(float a, float b) {
  union fi_t { float f; int32_t i; };
  fi_t fa, fb;
  fa.f = a; fb.f = b;
  return std::abs(fa.i - fb.i) <= FLOAT_ULP;
}

static void convolution_cpu(float *O, const float *I, const float *W, int32_t width, int32_t height) {
  int paddedWidth = width + 2;
  for (int32_t y = 0; y < height; ++y) {
    for (int32_t x = 0; x < width; ++x) {
      int paddedY = y + 1;
      int paddedX = x + 1;
      float sum = 0.0f;
      for (int32_t ky = -1; ky <= 1; ++ky) {
        for (int32_t kx = -1; kx <= 1; ++kx) {
          int32_t iy = paddedY + ky;
          int32_t ix = paddedX + kx;
          float value = I[iy * paddedWidth + ix];
          float weight = W[(ky + 1) * 3 + (kx + 1)];
          sum += value * weight;
        }
      }
      O[y * width + x] = sum;
    }
  }
}

__global__ void conv3(float* __restrict__ output,
                      const float* __restrict__ input,
                      const float* __restrict__ weights,
                      int width,
                      int height)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= width || y >= height) return;

  const int paddedWidth = width + 2;
  const int px = x + 1;
  const int py = y + 1;

  float sum = 0.0f;
  sum += input[(py - 1) * paddedWidth + (px - 1)] * weights[0];
  sum += input[(py - 1) * paddedWidth +  px      ] * weights[1];
  sum += input[(py - 1) * paddedWidth + (px + 1)] * weights[2];
  sum += input[ py      * paddedWidth + (px - 1)] * weights[3];
  sum += input[ py      * paddedWidth +  px      ] * weights[4];
  sum += input[ py      * paddedWidth + (px + 1)] * weights[5];
  sum += input[(py + 1) * paddedWidth + (px - 1)] * weights[6];
  sum += input[(py + 1) * paddedWidth +  px      ] * weights[7];
  sum += input[(py + 1) * paddedWidth + (px + 1)] * weights[8];

  output[y * width + x] = sum;
}

// --- Globals / CLI ---
static int g_size = 32;
static int g_bx = 1;
static int g_by = 1;

static float* d_i = nullptr;
static float* d_w = nullptr;
static float* d_o = nullptr;

static bool is_number(const char* s) {
  if (!s || !*s) return false;
  char* end = nullptr;
  long v = strtol(s, &end, 10);
  (void)v;
  return end && *end == '\0';
}

static void show_usage() {
  printf("Usage: [-n size] [-bx X] [-by Y] [-h]\n");
  printf("  examples:\n");
  printf("    ./prog -n 256 -bx 32 -by 16\n");
  printf("    ./prog -n 1024 -bx32 -by16\n");
}

static void parse_args(int argc, char** argv) {
  for (int i = 1; i < argc; ++i) {
    const char* a = argv[i];

    if (!strcmp(a, "-h") || !strcmp(a, "--help")) {
      show_usage(); exit(0);
    }
    else if (!strcmp(a, "-n")) {
      if (i + 1 >= argc || !is_number(argv[i+1])) {
        fprintf(stderr, "Error: -n requires an integer value\n");
        exit(1);
      }
      g_size = atoi(argv[++i]);
    }
    else if (!strcmp(a, "-bx")) {
      if (i + 1 >= argc || !is_number(argv[i+1])) {
        fprintf(stderr, "Error: -bx requires an integer value\n");
        exit(1);
      }
      g_bx = atoi(argv[++i]);
    }
    else if (!strcmp(a, "-by")) {
      if (i + 1 >= argc || !is_number(argv[i+1])) {
        fprintf(stderr, "Error: -by requires an integer value\n");
        exit(1);
      }
      g_by = atoi(argv[++i]);
    }
    // support attached forms: -bx32 / -by16
    else if (!strncmp(a, "-bx", 3) && is_number(a + 3)) {
      g_bx = atoi(a + 3);
    }
    else if (!strncmp(a, "-by", 3) && is_number(a + 3)) {
      g_by = atoi(a + 3);
    }
    else {
      fprintf(stderr, "Unknown option or bad value: %s\n", a);
      show_usage(); exit(1);
    }
  }

  // basic validation
  if (g_size <= 0 || g_bx <= 0 || g_by <= 0) {
    fprintf(stderr, "Error: size and block dims must be positive\n");
    exit(1);
  }

  printf("Parsed: size=%d, bx=%d, by=%d\n", g_size, g_bx, g_by);
}

static void cleanup() {
  if (d_i) { hipError_t e = hipFree(d_i); if (e != hipSuccess) std::fprintf(stderr, "cudaFree(d_i): %s\n", hipGetErrorString(e)); d_i = nullptr; }
  if (d_w) { hipError_t e = hipFree(d_w); if (e != hipSuccess) std::fprintf(stderr, "cudaFree(d_w): %s\n", hipGetErrorString(e)); d_w = nullptr; }
  if (d_o) { hipError_t e = hipFree(d_o); if (e != hipSuccess) std::fprintf(stderr, "cudaFree(d_o): %s\n", hipGetErrorString(e)); d_o = nullptr; }
  hipDeviceReset();
}

int main(int argc, char** argv) {
  parse_args(argc, argv);

  const int width  = g_size;
  const int height = g_size;
  const uint32_t o_points = (uint32_t)width * height;
  const uint32_t i_points = (uint32_t)(width + 2) * (height + 2);
  const uint32_t w_points = 9;

  std::printf("Create context\n");
  CUDA_CHECK(hipSetDevice(0));
  hipDeviceProp_t prop{}; CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
  std::printf("Using device: %s\n", prop.name);

  const size_t i_nbytes = i_points * sizeof(float);
  const size_t w_nbytes = w_points * sizeof(float);
  const size_t o_nbytes = o_points * sizeof(float);
  CUDA_CHECK(hipMalloc(&d_i, i_nbytes));
  CUDA_CHECK(hipMalloc(&d_w, w_nbytes));
  CUDA_CHECK(hipMalloc(&d_o, o_nbytes));

  std::vector<float> h_i(i_points);
  std::vector<float> h_w(w_points);
  std::vector<float> h_o(o_points, 0.0f);

  for (int32_t y = -1; y < height + 1; ++y) {
    for (int32_t x = -1; x < width + 1; ++x) {
      float v = (x >= 0 && x < width && y >= 0 && y < height)
                ? static_cast<float>(std::rand()) / static_cast<float>(RAND_MAX)
                : 0.0f;
      h_i[(y + 1) * (width + 2) + (x + 1)] = v;
    }
  }
  for (uint32_t i = 0; i < w_points; ++i)
    h_w[i] = static_cast<float>(std::rand()) / static_cast<float>(RAND_MAX);

  CUDA_CHECK(hipMemcpy(d_i, h_i.data(), i_nbytes, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_w, h_w.data(), w_nbytes, hipMemcpyHostToDevice));

  // Launch config: block=(bx,by), grid=(ceil(width/bx), ceil(height/by))
  dim3 block((unsigned)g_bx, (unsigned)g_by, 1);
  dim3 grid( (width  + g_bx - 1) / g_bx,
             (height + g_by - 1) / g_by,
             1 );
  std::printf("Launch grid=(%u,%u,1), block=(%u,%u,1)\n", grid.x, grid.y, block.x, block.y);

  auto t0 = std::chrono::high_resolution_clock::now();
  conv3<<<grid, block>>>(d_o, d_i, d_w, width, height);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
  auto t1 = std::chrono::high_resolution_clock::now();
  double elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(t1 - t0).count();
  std::printf("Elapsed time: %lg ms\n", elapsed);

  CUDA_CHECK(hipMemcpy(h_o.data(), d_o, o_nbytes, hipMemcpyDeviceToHost));

  std::vector<float> ref(o_points);
  convolution_cpu(ref.data(), h_i.data(), h_w.data(), width, height);

  int errors = 0;
  for (uint32_t i = 0; i < o_points; ++i) {
    if (!compare_equal(h_o[i], ref[i])) {
      if (errors < 100)
        std::printf("*** error: [%u] expected=%f, actual=%f\n", i, ref[i], h_o[i]);
      ++errors;
    }
  }
  if (errors) std::printf("FAILED! - %d errors\n", errors);
  else        std::printf("PASSED!\n");

  cleanup();
  return errors;
}
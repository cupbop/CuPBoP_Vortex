//-----------------------------------------------------------------------
// Reference
//
// Harris, M. and Garland, M., 2012.
// Optimizing parallel prefix operations for the Fermi architecture.
// In GPU Computing Gems Jade Edition (pp. 29-38). Morgan Kaufmann.
//-----------------------------------------------------------------------

#include <cstdio>
#include <cstring>
#include <chrono>
#include <hip/hip_runtime.h>

__device__ __inline__ unsigned int lanemask_lt()
{
#ifdef ASM
  unsigned int mask;
  asm("mov.u32 %0, %lanemask_lt;" : "=r"(mask));
  return mask;
#else
  const unsigned int lane = threadIdx.x & (warpSize-1);
  return (1 << (lane)) - 1;
#endif
}

// positive numbers
__host__ __device__ __inline__
bool valid(int x) {
  return x > 0;
}

__device__ __inline__ int block_binary_prefix_sums(int x,
                                                   int* __restrict__ blockCtr,
                                                   int* __restrict__ turnCtr)
{
  const int idx     = threadIdx.x;
  const int lane    = idx & (warpSize - 1);
  const int warpIdx = idx >> 5;

  const bool p = valid(x);
#if (CUDART_VERSION < 9000)
  const unsigned m = __ballot(p);
#else
  const unsigned m = __ballot_sync(0xFFFFFFFFu, p);
#endif
  const int rank       = __popc(m & lanemask_lt());
  const int warpCount  = __popc(m);

  // One atomicAdd per warp, executed in warpIdx order.
  int base = 0;
  if (lane == 0) {
    // spin until it's this warp's turn
    while (atomicAdd(turnCtr, 0) != warpIdx) { }
    base = atomicAdd(blockCtr, warpCount);   // reserve a contiguous block
    atomicAdd(turnCtr, 1);                   // allow next warp
  }
#if (CUDART_VERSION < 9000)
  base = __shfl(base, 0);
#else
  base = __shfl_sync(0xFFFFFFFFu, base, 0);
#endif

  return base + rank; 
}

__global__ void binary_scan(
        int *__restrict__ g_odata,
  const int *__restrict__ g_idata,
        int *__restrict__ blockCtr,
        int *__restrict__ turnCtr)
{
  int i = threadIdx.x;
  g_odata[i] = block_binary_prefix_sums(g_idata[i], blockCtr, turnCtr);
}

template <int N>
void bscan (const int repeat) 
{
  int h_in[N];
  int h_out[N];
  int ref_out[N];

  int *d_in, *d_out;

  // two small per-launch device scalars (per-block counters)
  int *d_blockCtr, *d_turnCtr;

  hipMalloc((void**)&d_in, N*sizeof(int));
  hipMalloc((void**)&d_out, N*sizeof(int));
  hipMalloc((void**)&d_blockCtr, sizeof(int));
  hipMalloc((void**)&d_turnCtr, sizeof(int));

  bool ok = true;
  double time = 0.0;
  srand(123);

  // size_t grid_size = 12*7*8*9*10;
  size_t grid_size = 1;
  dim3 grids (grid_size);
  dim3 blocks (N);

  int valid_count = 0;

  for (int i = 0; i < repeat; i++) {
    for (int n = 0; n < N; n++) {
      h_in[n] = rand() % N - N/2;
      if (valid(h_in[n])) valid_count++;  // total number of valid elements
    }
    hipMemcpy(d_in, h_in, N*sizeof(int), hipMemcpyHostToDevice);

    // reset per-block counters
    hipMemset(d_blockCtr, 0, sizeof(int));
    hipMemset(d_turnCtr,  0, sizeof(int));

    hipDeviceSynchronize();
    auto start = std::chrono::steady_clock::now();

    binary_scan<<<grids, blocks>>>(d_out, d_in, d_blockCtr, d_turnCtr);

    hipDeviceSynchronize();
    auto end = std::chrono::steady_clock::now();
    time += std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();

    // verify exclusive sum
    hipMemcpy(h_out, d_out, N*sizeof(int), hipMemcpyDeviceToHost);

    ref_out[0] = 0;
    ok &= (h_out[0] == ref_out[0]);
    for (int i = 1; i < N; i++) {
      ref_out[i] = ref_out[i-1] + (h_in[i-1] > 0);
      ok &= (ref_out[i] == h_out[i]);
      // printf("i=%d ref=%d out=%d in=%d\n", i, ref_out[i], h_out[i], h_in[i]);
    }
    if (!ok) break;
  } // for

  printf("Block size = %d, ratio of valid elements = %f, verify = %s\n",
         N, valid_count * 1.f / (N * repeat), ok ? "PASS" : "FAIL");

  if (ok) {
    printf("Average execution time: %f (us)\n", (time * 1e-3f) / repeat);
    printf("Billion elements per second: %f\n\n",
            grid_size * N * repeat / time);
  }

  hipFree(d_in);
  hipFree(d_out);
  hipFree(d_blockCtr);
  hipFree(d_turnCtr);
}

int main(int argc, char* argv[])
{
  if (argc != 2) {
    printf("Usage: %s <repeat>\n", argv[0]);
    return 1;
  }
  const int repeat = atoi(argv[1]);

  // scan over N elements (N = [32, 1024])
  bscan<32>(repeat);
  bscan<64>(repeat);
  bscan<128>(repeat);
  bscan<256>(repeat);
  bscan<512>(repeat);

  return 0;
}

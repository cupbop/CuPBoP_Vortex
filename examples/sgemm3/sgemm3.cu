// sgemm3.cu
// nvcc -O2 -std=c++17 sgemm3.cu -o sgemm3
#include <cstdio>
#include <cstdlib>
#include <cstdint>
#include <cassert>
#include <cstring>
#include <cmath>
#include <vector>
#include <chrono>
#include <unistd.h>     // getopt
#include <hip/hip_runtime.h>

// ===== [common.h] 통합 =====
#ifndef TYPE
#define TYPE float
#endif
// ===========================
 
#define FLOAT_ULP 6

#define CUDA_CHECK(expr)                                                         \
  do {                                                                           \
    hipError_t _err = (expr);                                                   \
    if (_err != hipSuccess) {                                                   \
      std::fprintf(stderr, "CUDA Error: %s -> %d (%s)\n", #expr, (int)_err,      \
                   hipGetErrorString(_err));                                    \
      std::exit(-1);                                                             \
    }                                                                            \
  } while (0)

// ---- OpenCL 호스트 코드와 동등한 Comparator/검증 유틸 ----
template <typename T> struct Comparator {};

template <> struct Comparator<int> {
  static const char* type_str() { return "integer"; }
  static int generate() { return std::rand(); }
  static bool compare(int a, int b, int index, int errors) {
    if (a != b) {
      if (errors < 100)
        std::printf("*** error: [%d] expected=%d, actual=%d\n", index, a, b);
      return false;
    }
    return true;
  }
};

template <> struct Comparator<float> {
  static const char* type_str() { return "float"; }
  static float generate() { return static_cast<float>(std::rand()) / RAND_MAX; }
  static bool compare(float a, float b, int index, int errors) {
    union fi_t { float f; int32_t i; };
    fi_t fa, fb; fa.f = a; fb.f = b;
    auto d = std::abs(fa.i - fb.i);
    if (d > FLOAT_ULP) {
      if (errors < 100)
        std::printf("*** error: [%d] expected=%f, actual=%f\n", index, a, b);
      return false;
    }
    return true;
  }
};

// ---- CPU ref (OpenCL 호스트의 sgemm_cpu와 동일) ----
// 주의: column-major 인덱싱 (C[n*M+m] = Σ A[k*M+m]*B[n*K+k])
static void sgemm_cpu(TYPE *C, const TYPE* A, const TYPE *B, int M, int N, int K) {
  for (int m = 0; m < M; ++m) {
    for (int n = 0; n < N; ++n) {
      TYPE acc = 0;
      for (int k = 0; k < K; ++k) {
        acc += A[k * M + m] * B[n * K + k];
      }
      C[n * M + m] = acc;
    }
  }
}

// ===== CUDA Kernel (OpenCL sgemm3와 1:1 논리) =====
// OpenCL 커널의 __local TYPE *localA, *localB 를 CUDA에선
// dynamic shared memory로 합쳐서 할당 -> [localA | localB]
__global__ void sgemm3_kernel(const TYPE* __restrict__ A,
                              const TYPE* __restrict__ B,
                              TYPE* __restrict__ C,
                              unsigned int N)
{
  // 스레드/블록 인덱스 매핑 (OpenCL과 동일 의미)
  int globalCol = blockIdx.x * blockDim.x + threadIdx.x; // get_global_id(0)
  int globalRow = blockIdx.y * blockDim.y + threadIdx.y; // get_global_id(1)
  int localCol  = threadIdx.x;                           // get_local_id(0)
  int localRow  = threadIdx.y;                           // get_local_id(1)
  int localSize = blockDim.x;                            // get_local_size(0) == tile_size

  extern __shared__ TYPE smem[];
  TYPE* localA = smem;                                   // size: localSize*localSize
  TYPE* localB = smem + localSize * localSize;           // size: localSize*localSize

  TYPE sum = 0;

  // for (int k = 0; k < N; k += localSize)
  for (int k = 0; k < (int)N; k += localSize) {
    // localA[localRow * localSize + localCol] = A[globalRow * N + (k + localCol)];
    // localB[localRow * localSize + localCol] = B[(k + localRow) * N + globalCol];
    localA[localRow * localSize + localCol] = A[(k + localCol) * N + globalRow];
    localB[localRow * localSize + localCol] = B[globalCol * N + (k + localRow)];

    __syncthreads(); // barrier(CLK_LOCAL_MEM_FENCE)

    // for (int j = 0; j < localSize; j++)
    //   sum += localA[localRow*localSize + j] * localB[j*localSize + localCol];
    for (int j = 0; j < localSize; ++j) {
      sum += localA[localRow * localSize + j] * localB[j * localSize + localCol];
    }

    __syncthreads(); // barrier(CLK_LOCAL_MEM_FENCE)
  }

  // C[globalRow * N + globalCol] = sum;   (row-major 스타일, OpenCL 커널과 동일)
  C[globalCol * N + globalRow] = sum;
}

// ===================== Host (OpenCL 호스트 로직 매핑) =====================
int main(int argc, char** argv) {
  std::srand(0xC0FFEE);

  uint32_t size = 16;      // N
  uint32_t tile_size = 4;  // local_size

  // 옵션 파싱: -n size, -t tile_size
  int opt;
  while ((opt = ::getopt(argc, argv, "n:t:h")) != -1) {
    switch (opt) {
      case 'n': size = std::max(1, atoi(optarg)); break;
      case 't': tile_size = std::max(1, atoi(optarg)); break;
      case 'h':
      default:
        std::printf("Usage: [-n size] [-t tile_size] [-h]\n");
        return 0;
    }
  }

  uint32_t size_sq = size * size;

  std::printf("Matrix size=%ux%u, tile size=%ux%u\n", size, size, tile_size, tile_size);
  if ((size / tile_size) * tile_size != size) {
    std::printf("Error: matrix size must be a multiple of %u\n", tile_size);
    return -1;
  }

  // 디바이스 정보
  int dev = 0;
  CUDA_CHECK(hipGetDevice(&dev));
  hipDeviceProp_t prop{};
  CUDA_CHECK(hipGetDeviceProperties(&prop, dev));
  std::printf("Using device: %s\n", prop.name);

  // 호스트 버퍼
  std::vector<TYPE> h_a(size_sq), h_b(size_sq), h_c(size_sq);

  for (uint32_t i = 0; i < size_sq; ++i) {
    h_a[i] = Comparator<TYPE>::generate();
    h_b[i] = Comparator<TYPE>::generate();
  }

  // 디바이스 버퍼
  TYPE *d_a = nullptr, *d_b = nullptr, *d_c = nullptr;
  size_t nbytes = size_sq * sizeof(TYPE);
  CUDA_CHECK(hipMalloc((void**)&d_a, nbytes));
  CUDA_CHECK(hipMalloc((void**)&d_b, nbytes));
  CUDA_CHECK(hipMalloc((void**)&d_c, nbytes));

  // 업로드
  std::printf("Upload source buffers\n");
  CUDA_CHECK(hipMemcpy(d_a, h_a.data(), nbytes, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_b, h_b.data(), nbytes, hipMemcpyHostToDevice));

  // 런치 파라미터 (OpenCL: global={size,size}, local={tile_size,tile_size})
  dim3 block(tile_size, tile_size, 1);
  dim3 grid (size / tile_size, size / tile_size, 1);
  std::printf("global_size=%u, %u\n", size, size);
  std::printf("local_size=%u, %u\n", tile_size, tile_size);

  // Dynamic shared memory: localA + localB
  size_t smem_bytes = 2ull * tile_size * tile_size * sizeof(TYPE);

  // 실행 & 타이밍
  auto t0 = std::chrono::high_resolution_clock::now();
  sgemm3_kernel<<<grid, block, smem_bytes>>>(d_a, d_b, d_c, size);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
  auto t1 = std::chrono::high_resolution_clock::now();
  double elapsed_ms = std::chrono::duration_cast<std::chrono::milliseconds>(t1 - t0).count();
  std::printf("Elapsed time: %lg ms\n", elapsed_ms);

  // 다운로드
  std::printf("Download destination buffer\n");
  CUDA_CHECK(hipMemcpy(h_c.data(), d_c, nbytes, hipMemcpyDeviceToHost));

  // 검증 (OpenCL 호스트와 동일 sgemm_cpu 사용 — column-major ref)
  std::printf("Verify result\n");
  std::vector<TYPE> h_ref(size_sq);
  sgemm_cpu(h_ref.data(), h_a.data(), h_b.data(), size, size, size);
  int errors = 0;
  for (uint32_t i = 0; i < size_sq; ++i) {
    if (!Comparator<TYPE>::compare(h_c[i], h_ref[i], i, errors)) {
      ++errors;
    }
  }
  if (errors != 0) {
    std::printf("FAILED! - %d errors\n", errors);
  } else {
    std::printf("PASSED!\n");
  }

  // 정리
  CUDA_CHECK(hipFree(d_a));
  CUDA_CHECK(hipFree(d_b));
  CUDA_CHECK(hipFree(d_c));
  return errors;
}
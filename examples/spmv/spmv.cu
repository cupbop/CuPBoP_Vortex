// spmv_jds_cuda.cu — CUDA JDS SpMV (OpenCL Parboil 스타일 1:1 대응)
// Build: nvcc -O3 -std=c++17 -arch=sm_70 -o spmv_jds_cuda spmv_jds_cuda.cu
// Run  : ./spmv_jds_cuda -i matrix.mtx,vector.vec [-o out.bin] [-p 32]
#include <hip/hip_runtime.h>

#include <cstdio>
#include <cstdlib>
#include <cstdint>
#include <cstring>
#include <string>
#include <vector>
#include <fstream>
#include <sstream>
#include <algorithm>
#include <cmath>
#include <unistd.h>  // getopt
#include <chrono>

#ifndef TYPE
#define TYPE float
#endif

#define CUDA_CHECK(expr) do {                                         \
  hipError_t _err = (expr);                                          \
  if (_err != hipSuccess) {                                          \
    fprintf(stderr, "CUDA Error: %s => %d (%s) at %s:%d\n",           \
            #expr, (int)_err, hipGetErrorString(_err),               \
            __FILE__, __LINE__);                                      \
    std::abort();                                                     \
  }                                                                   \
} while (0)

// ------------------------ 유틸/입출력 ------------------------
static void usage(const char* prog){
  std::printf("Usage: %s -i <matrix.mtx,vector.vec> [-o out.bin] [-p pad]\n", prog);
}

static std::string replace_ext(const std::string& path, const std::string& ext){
  auto p = path.find_last_of('.');
  if (p == std::string::npos) return path + ext;
  return path.substr(0, p) + ext;
}

// gold: (int N; float N개) 바이너리
static float* read_gold_file(const char* filename, int* outN){
  FILE* f = std::fopen(filename, "rb");
  if (!f) return nullptr;
  int N=0;
  if (std::fread(&N, sizeof(int), 1, f) != 1){ std::fclose(f); return nullptr; }
  float* buf = (float*)std::malloc(sizeof(float)*N);
  if (!buf){ std::fclose(f); return nullptr; }
  if ((int)std::fread(buf, sizeof(float), N, f) != N){
    std::free(buf); std::fclose(f); return nullptr;
  }
  std::fclose(f);
  if (outN) *outN = N;
  return buf;
}

static int write_bin_with_size(const char* filename, const float* y, int N){
  FILE* f = std::fopen(filename, "wb");
  if (!f){ perror("fopen"); return -1; }
  if (std::fwrite(&N, sizeof(int), 1, f) != 1){ std::fclose(f); return -1; }
  if ((int)std::fwrite(y, sizeof(float), N, f) != N){ std::fclose(f); return -1; }
  std::fclose(f);
  return 0;
}

static int compare_floats(const float* src, const float* gold, int count){
  int num_errors = 0;
  float max_abs = 0.0f;
  for (int i=0;i<count;++i) max_abs = std::max(max_abs, std::fabs(gold[i]));
  float abstol = 1e-4f * max_abs; // 원본 로직
  for (int i=0;i<count;++i){
    float diff = std::fabs(gold[i] - src[i]);
    if (!(diff <= abstol || diff < 0.002f * std::fabs(gold[i]))){
      if (num_errors < 10)
        std::printf("Fail at row %d: (gold) %f != %f (computed)\n", i, gold[i], src[i]);
      ++num_errors;
    }
  }
  return num_errors;
}

// 텍스트/MatrixMarket/바이너리 모두 지원
static bool read_vec_auto(const std::string& path, std::vector<TYPE>& x, int expectedN){
  // 1) 텍스트 시도
  {
    std::ifstream ifs(path);
    if (ifs) {
      x.clear(); x.reserve(expectedN);
      double v;
      while (ifs >> v) x.push_back((TYPE)v);
      if ((int)x.size() == expectedN) return true;
    }
  }

  // 2) MatrixMarket 텍스트 시도
  {
    std::ifstream ifs(path);
    if (ifs) {
      std::string line;
      if (std::getline(ifs, line) && line.rfind("%%MatrixMarket", 0) == 0) {
        // 사이즈 라인까지 스킵
        while (std::getline(ifs, line)) {
          if (!line.empty() && line[0] != '%') break;
        }
        if (!line.empty()) {
          std::istringstream ss(line);
          int n=0, m=1; // "n" 또는 "n m" 형태 모두 수용
          ss >> n; if (ss >> m) { /* ok */ }
          // 배열(array) 스타일: 값 n개
          x.clear(); x.reserve(n);
          double v;
          while (ifs >> v) x.push_back((TYPE)v);
          if ((int)x.size()==n && n==expectedN) return true;
        }
      }
    }
  }

  // 파일 크기 얻기
  auto file_size = [](const std::string& p)->long long {
    std::ifstream f(p, std::ios::binary | std::ios::ate);
    if (!f) return -1;
    return (long long)f.tellg();
  };
  long long sz = file_size(path);

  // 3) 바이너리: int N + N floats
  {
    std::ifstream ifs(path, std::ios::binary);
    if (ifs) {
      int N = 0;
      ifs.read(reinterpret_cast<char*>(&N), sizeof(int));
      if (ifs && N > 0) {
        // 크기 검증(가능하면)
        if (sz < 0 || sz == (long long)sizeof(int) + (long long)N * sizeof(float) || N==expectedN) {
          std::vector<float> buf(N);
          ifs.read(reinterpret_cast<char*>(buf.data()), (std::streamsize)N*sizeof(float));
          if (ifs) {
            x.resize(N);
            for (int i=0;i<N;++i) x[i]=(TYPE)buf[i];
            return N==expectedN; // 기대 길이와 맞아야 성공으로 간주
          }
        }
      }
    }
  }

  // 4) 바이너리: N floats (헤더 없음)
  if (sz >= 0 && sz % (long long)sizeof(float) == 0) {
    int N = (int)(sz / (long long)sizeof(float));
    if (N == expectedN) {
      std::ifstream ifs(path, std::ios::binary);
      if (ifs) {
        std::vector<float> buf(N);
        ifs.read(reinterpret_cast<char*>(buf.data()), (std::streamsize)N*sizeof(float));
        if (ifs) {
          x.resize(N);
          for (int i=0;i<N;++i) x[i]=(TYPE)buf[i];
          return true;
        }
      }
    }
  }

  return false;
}

// ------------------------ Matrix Market -> COO ------------------------
struct COO {
  int rows=0, cols=0;
  std::vector<int>   I, J;
  std::vector<TYPE>  V;
};

static bool read_matrix_market(const std::string& path, COO& coo){
  std::ifstream ifs(path);
  if (!ifs){
    std::fprintf(stderr, "Cannot open %s\n", path.c_str());
    return false;
  }
  std::string header;
  if (!std::getline(ifs, header)) return false;
  if (header.rfind("%%MatrixMarket", 0) != 0){
    std::fprintf(stderr, "Not a MatrixMarket file\n");
    return false;
  }

  // 헤더 토큰 파싱: coordinate / array, real|integer|pattern, general|symmetric 등
  // ex) "%%MatrixMarket matrix coordinate real symmetric"
  bool is_coordinate = header.find("coordinate") != std::string::npos;
  bool is_symmetric  = header.find("symmetric")  != std::string::npos;

  // 사이즈 라인까지 주석 스킵
  std::string line;
  do {
    if (!std::getline(ifs, line)) return false;
  } while (line.empty() || line[0] == '%');

  int M,N,NNZ;
  {
    std::istringstream ss(line);
    if (!(ss >> M >> N >> NNZ)){
      std::fprintf(stderr,"Bad size line\n");
      return false;
    }
  }

  coo.rows=M; coo.cols=N;
  coo.I.reserve(is_symmetric ? NNZ*2 : NNZ);
  coo.J.reserve(is_symmetric ? NNZ*2 : NNZ);
  coo.V.reserve(is_symmetric ? NNZ*2 : NNZ);

  if (!is_coordinate){
    std::fprintf(stderr,"Only coordinate format is supported\n");
    return false;
  }

  for (int k=0;k<NNZ;++k){
    if (!std::getline(ifs, line)) return false;
    if (line.empty() || line[0]=='%'){ --k; continue; }
    std::istringstream t(line);
    int i,j; double v;
    if (!(t >> i >> j >> v)) return false;
    --i; --j;                // 0-based

    coo.I.push_back(i);
    coo.J.push_back(j);
    coo.V.push_back((TYPE)v);

    if (is_symmetric && i!=j){
      coo.I.push_back(j);
      coo.J.push_back(i);
      coo.V.push_back((TYPE)v);
    }
  }
  return true;
}

// ------------------------ COO -> CSR ------------------------
struct CSR {
  int rows=0, cols=0;
  std::vector<int>   Ap; // size rows+1
  std::vector<int>   Aj; // size nnz
  std::vector<TYPE>  Ax; // size nnz
};

static void coo_to_csr(const COO& coo, CSR& csr){
  csr.rows = coo.rows; csr.cols = coo.cols;
  int M = csr.rows; size_t NNZ = coo.I.size();
  csr.Ap.assign(M+1, 0);
  csr.Aj.resize(NNZ);
  csr.Ax.resize(NNZ);

  // row counts
  for (size_t k=0;k<NNZ;++k) ++csr.Ap[coo.I[k]];
  // prefix
  int sum=0;
  for (int r=0;r<M;++r){ int c=csr.Ap[r]; csr.Ap[r]=sum; sum+=c; }
  csr.Ap[M]=sum;

  // scatter (stable)
  std::vector<int> next(csr.Ap.begin(), csr.Ap.end());
  for (size_t k=0;k<NNZ;++k){
    int r = coo.I[k];
    int p = next[r]++;
    csr.Aj[p] = coo.J[k];
    csr.Ax[p] = coo.V[k];
  }

  // 각 행 안에서 column 정렬(선택) — 안정성을 위해
  for (int r=0;r<M;++r){
    int s=csr.Ap[r], e=csr.Ap[r+1], len=e-s;
    std::vector<int> idx(len);
    for (int t=0;t<len;++t) idx[t]=s+t;
    std::sort(idx.begin(), idx.end(), [&](int a,int b){ return csr.Aj[a]<csr.Aj[b]; });
    std::vector<int>   newJ; newJ.reserve(len);
    std::vector<TYPE>  newV; newV.reserve(len);
    for (int t=0;t<len;++t){ newJ.push_back(csr.Aj[idx[t]]); newV.push_back(csr.Ax[idx[t]]); }
    for (int t=0;t<len;++t){ csr.Aj[s+t]=newJ[t]; csr.Ax[s+t]=newV[t]; }
  }
}

// ------------------------ CSR -> JDS ------------------------
struct JDS {
  // JDS 배열(Parboil 스타일)
  // data/indices : 대각(diagonal)별 연속 저장, 각 대각 시작은 jds_ptr[k]
  // perm        : 행 재배열(원래 행 idx)
  // sh_zcnt     : 블록별 유효 대각 개수 (크기 = numBlocks = ceil(dim/pad))
  // depth       : 대각 수 = max row nnz
  // len         : data/indices 길이(=nnz after sort)
  int dim=0, depth=0, len=0, nzcnt_len=0; // nzcnt_len = numBlocks
  std::vector<TYPE> data;
  std::vector<int>  indices;
  std::vector<int>  perm;
  std::vector<int>  jds_ptr;   // size = depth
  std::vector<int>  sh_zcnt;   // size = nzcnt_len
};

static void csr_to_jds(const CSR& csr, int pad, JDS& jds){
  const int M = csr.rows;
  jds.dim = M;

  // 1) 행별 nnz와 depth
  std::vector<int> row_nnz(M);
  int depth = 0;
  for (int r=0;r<M;++r){
    row_nnz[r] = csr.Ap[r+1]-csr.Ap[r];
    depth = std::max(depth, row_nnz[r]);
  }
  jds.depth = depth;

  // 2) perm: nnz 내림차순
  jds.perm.resize(M);
  for (int r=0;r<M;++r) jds.perm[r]=r;
  std::stable_sort(jds.perm.begin(), jds.perm.end(),
                   [&](int a,int b){ return row_nnz[a] > row_nnz[b]; });

  // 3) 각 대각의 실제 길이 diag_len[k]
  std::vector<int> diag_len(depth, 0);
  for (int k=0;k<depth;++k){
    int cnt=0;
    for (int p=0;p<M;++p){
      int r = jds.perm[p];
      if (row_nnz[r] > k) ++cnt; else break;
    }
    diag_len[k]=cnt;
  }

  // 4) 패딩된 길이와 jds_ptr
  std::vector<int> diag_len_padded(depth, 0);
  jds.jds_ptr.resize(depth);
  int off = 0;
  for (int k=0;k<depth;++k){
    int L  = diag_len[k];
    int LP = (L + pad - 1) / pad * pad;     // pad 배수로
    diag_len_padded[k] = LP;
    jds.jds_ptr[k] = off;
    off += LP;
  }
  jds.len = off;

  jds.data.assign(jds.len, (TYPE)0);
  jds.indices.assign(jds.len, 0);

  // 5) 데이터 채우기 (유효 구간만 채우고 나머지는 0으로 유지)
  for (int p=0; p<M; ++p){
    int r = jds.perm[p];
    int s = csr.Ap[r], e = csr.Ap[r+1], nnz = e - s;
    for (int k=0;k<nnz;++k){
      int base = jds.jds_ptr[k];
      // k번째 대각에서 p번째 위치는 항상 존재 (패딩 덕분)
      int j = base + p;
      jds.data[j]   = csr.Ax[s+k];
      jds.indices[j]= csr.Aj[s+k];
    }
  }

  // 6) sh_zcnt (블록 그룹별 유효 대각 수)
  int numBlocks = (M + pad - 1) / pad;
  jds.nzcnt_len = numBlocks;
  jds.sh_zcnt.assign(numBlocks, 0);
  for (int b=0;b<numBlocks;++b){
    int row_start = b*pad;
    int bound=0;
    for (int k=0;k<depth;++k){
      if (row_start < diag_len[k]) ++bound; else break;
    }
    jds.sh_zcnt[b]=bound;
  }
}

// ------------------------ CPU 참조 (검증용) ------------------------
static void spmv_jds_cpu(const JDS& jds, const std::vector<TYPE>& x, std::vector<TYPE>& y){
  y.assign(jds.dim, (TYPE)0);
  // 원본 Parboil 커널과 동일한 수식:
  // 각 permuted row i에 대해 sum_k data[jds_ptr[k]+i] * x[indices[jds_ptr[k]+i]]
  // 결과는 원래 행 위치 d_perm[i]에 저장
  for (int i=0;i<jds.dim;++i){
    TYPE sum=0;
    int bound = jds.depth; // 실제로는 유효 대각까지만 보지만, out-of-range는 diag_len으로 비어있음
    for (int k=0;k<bound;++k){
      int j = jds.jds_ptr[k] + i;
      if (j >= jds.len) break; // 안전 가드
      // 유효한지 확인: i가 이 대각의 길이보다 작은지? (j 인덱스가 data 범위 내인지로 충분)
      sum += jds.data[j] * x[jds.indices[j]];
    }
    // perm 역매핑: perm[p]=원행, 여기선 p=i의 원행
    y[jds.perm[i]] = sum;
  }
}

// ------------------------ CUDA 커널 ------------------------
__global__ void spmv_jds_naive_kernel(
    float* __restrict__ d_Ax,
    const float* __restrict__ d_data,
    const int*   __restrict__ d_idx,
    const int*   __restrict__ d_perm,
    const float* __restrict__ d_x,
    int dim,
    const int*   __restrict__ jds_ptr,
    const int*   __restrict__ sh_zcnt,
    int pad)                             // ← pad 추가
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= dim) return;

  int group = i / pad;                   // ← OpenCL의 ix/32와 동일 개념
  float sum = 0.0f;
  int bound = sh_zcnt[group];            // ← 여기!

  #pragma unroll 1
  for (int k = 0; k < bound; ++k) {
    int j = jds_ptr[k] + i;              // 패딩이 되어 있어야 안전 (수정 2 참고)
    sum += d_data[j] * d_x[d_idx[j]];
  }
  d_Ax[d_perm[i]] = sum;
}

// ------------------------ main ------------------------
int main(int argc, char** argv){
  std::string in_pair; // "matrix.mtx,vector.vec"
  std::string out_path;
  int pad = 32;

  int opt;
  while ((opt = getopt(argc, argv, "i:o:p:h")) != -1){
    switch(opt){
      case 'i': in_pair = optarg; break;
      case 'o': out_path = optarg; break;
      case 'p': pad = std::max(1, std::atoi(optarg)); break;
      case 'h': usage(argv[0]); return 0;
      default : usage(argv[0]); return 1;
    }
  }
  if (in_pair.empty()){
    usage(argv[0]); return 1;
  }

  // "mtx,vec" 파싱
  std::string mtx_path, vec_path;
  {
    auto pos = in_pair.find(',');
    if (pos == std::string::npos){
      std::fprintf(stderr, "Expecting -i <matrix.mtx,vector.vec>\n");
      return 1;
    }
    mtx_path = in_pair.substr(0,pos);
    vec_path = in_pair.substr(pos+1);
  }

  // Matrix Market 로드 -> COO -> CSR -> JDS
  COO coo;
  if (!read_matrix_market(mtx_path, coo)) return 1;
  CSR csr; coo_to_csr(coo, csr);
  if (csr.rows != csr.cols){
    std::fprintf(stderr,"Warning: non-square matrix %dx%d; continuing.\n", csr.rows, csr.cols);
  }
  JDS jds; csr_to_jds(csr, pad, jds);

  // x 벡터 로드 (크기 csr.cols)
  std::vector<TYPE> h_x(csr.cols);
  if (!read_vec_auto(vec_path, h_x, csr.cols)) {
    std::fprintf(stderr, "Failed to read vector '%s' with expected length %d\n",
                vec_path.c_str(), csr.cols);
    return 1;
  }

  // 디바이스 버퍼
  float *d_Ax=nullptr, *d_data=nullptr, *d_x=nullptr;
  int   *d_idx=nullptr, *d_perm=nullptr, *d_jds_ptr=nullptr, *d_shzcnt=nullptr;

  CUDA_CHECK(hipMalloc(&d_Ax,      sizeof(float)*jds.dim));
  CUDA_CHECK(hipMalloc(&d_data,    sizeof(float)*jds.len));
  CUDA_CHECK(hipMalloc(&d_idx,     sizeof(int)  *jds.len));
  CUDA_CHECK(hipMalloc(&d_perm,    sizeof(int)  *jds.dim));
  CUDA_CHECK(hipMalloc(&d_x,       sizeof(float)*csr.cols));
  CUDA_CHECK(hipMalloc(&d_jds_ptr, sizeof(int)  *jds.depth));
  CUDA_CHECK(hipMalloc(&d_shzcnt,  sizeof(int)  *jds.nzcnt_len));

  CUDA_CHECK(hipMemcpy(d_data,    jds.data.data(),   sizeof(float)*jds.len,      hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_idx,     jds.indices.data(),sizeof(int)  *jds.len,      hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_perm,    jds.perm.data(),   sizeof(int)  *jds.dim,      hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_x,       h_x.data(),        sizeof(float)*csr.cols,     hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_jds_ptr, jds.jds_ptr.data(),sizeof(int)  *jds.depth,    hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_shzcnt,  jds.sh_zcnt.data(),sizeof(int)  *jds.nzcnt_len,hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemset(d_Ax, 0, sizeof(float)*jds.dim));

  // 런치 구성: 원본과 동일 개념 — blockDim.x = pad, gridDim.x = ceil(dim/pad)
  dim3 block(pad, 1, 1);
  dim3 grid((jds.dim + pad - 1) / pad, 1, 1);

  std::printf("grid=%d, block=%d, dim=%d, depth=%d, len=%d\n",
              grid.x, block.x, jds.dim, jds.depth, jds.len);

  auto t0 = std::chrono::high_resolution_clock::now();
  spmv_jds_naive_kernel<<<grid, block>>>(d_Ax, d_data, d_idx, d_perm, d_x,
                                         jds.dim, d_jds_ptr, d_shzcnt, pad);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
  auto t1 = std::chrono::high_resolution_clock::now();
  double ms = std::chrono::duration<double, std::milli>(t1 - t0).count();
  std::printf("Elapsed: %.3f ms\n", ms);

  // 결과 수거
  std::vector<float> h_Ax(jds.dim);
  CUDA_CHECK(hipMemcpy(h_Ax.data(), d_Ax, sizeof(float)*jds.dim, hipMemcpyDeviceToHost));

  // out.bin 요청시 저장 (Parboil 포맷과 동일: int N + float[N])
  if (!out_path.empty()){
    if (write_bin_with_size(out_path.c_str(), h_Ax.data(), jds.dim) != 0){
      std::fprintf(stderr, "Failed to write %s\n", out_path.c_str());
    } else {
      std::printf("Wrote %s\n", out_path.c_str());
    }
  }

  // gold 비교 (matrix.mtx -> matrix.gold)
  {
    std::string gold_path = replace_ext(mtx_path, ".gold");
    int gN=0; float* gbuf = read_gold_file(gold_path.c_str(), &gN);
    if (gbuf){
      if (gN != jds.dim){
        std::printf("error: gold data size mismatch: current=%d, expected=%d\n", jds.dim, gN);
      } else {
        int errors = compare_floats(h_Ax.data(), gbuf, gN);
        if (errors > 0) std::printf("FAILED!\n"); else std::printf("PASSED!\n");
      }
      std::free(gbuf);
    } else {
      std::printf("(gold '%s' not found — skipping check)\n", gold_path.c_str());
    }
  }

  // 정리
  hipFree(d_Ax); hipFree(d_data); hipFree(d_idx);
  hipFree(d_perm); hipFree(d_x);
  hipFree(d_jds_ptr); hipFree(d_shzcnt);
  return 0;
}
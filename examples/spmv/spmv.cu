// spmv_jds_cuda.cu — CUDA JDS SpMV (OpenCL Parboil 스타일 1:1 대응)
// Build: nvcc -O3 -std=c++17 -arch=sm_70 -o spmv_jds_cuda spmv_jds_cuda.cu
// Run  : ./spmv_jds_cuda -i matrix.mtx,vector.vec [-o out.bin] [-p 32]
#include <hip/hip_runtime.h>

#include <cstdio>
#include <cstdlib>
#include <cstdint>
#include <cstring>
#include <string>
#include <vector>
#include <fstream>
#include <sstream>
#include <algorithm>
#include <cmath>
#include <unistd.h>  // getopt
#include <chrono>

#ifndef TYPE
#define TYPE float
#endif

#define CUDA_CHECK(expr) do {                                         \
  hipError_t _err = (expr);                                          \
  if (_err != hipSuccess) {                                          \
    fprintf(stderr, "CUDA Error: %s => %d (%s) at %s:%d\n",           \
            #expr, (int)_err, hipGetErrorString(_err),               \
            __FILE__, __LINE__);                                      \
    std::abort();                                                     \
  }                                                                   \
} while (0)

// ------------------------ 유틸/입출력 ------------------------
static void usage(const char* prog){
  std::printf("Usage: %s -i <matrix.mtx,vector.vec> [-o out.bin] [-p pad]\n", prog);
}

static std::string replace_ext(const std::string& path, const std::string& ext){
  auto p = path.find_last_of('.');
  if (p == std::string::npos) return path + ext;
  return path.substr(0, p) + ext;
}

// gold: (int N; float N개) 바이너리
static float* read_gold_file(const char* filename, int* outN){
  FILE* f = std::fopen(filename, "rb");
  if (!f) return nullptr;
  int N=0;
  if (std::fread(&N, sizeof(int), 1, f) != 1){ std::fclose(f); return nullptr; }
  float* buf = (float*)std::malloc(sizeof(float)*N);
  if (!buf){ std::fclose(f); return nullptr; }
  if ((int)std::fread(buf, sizeof(float), N, f) != N){
    std::free(buf); std::fclose(f); return nullptr;
  }
  std::fclose(f);
  if (outN) *outN = N;
  return buf;
}

static int write_bin_with_size(const char* filename, const float* y, int N){
  FILE* f = std::fopen(filename, "wb");
  if (!f){ perror("fopen"); return -1; }
  if (std::fwrite(&N, sizeof(int), 1, f) != 1){ std::fclose(f); return -1; }
  if ((int)std::fwrite(y, sizeof(float), N, f) != N){ std::fclose(f); return -1; }
  std::fclose(f);
  return 0;
}

static int compare_floats(const float* src, const float* gold, int count){
  int num_errors = 0;
  float max_abs = 0.0f;
  for (int i=0;i<count;++i) max_abs = std::max(max_abs, std::fabs(gold[i]));
  float abstol = 1e-4f * max_abs; // 원본 로직
  for (int i=0;i<count;++i){
    float diff = std::fabs(gold[i] - src[i]);
    if (!(diff <= abstol || diff < 0.002f * std::fabs(gold[i]))){
      if (num_errors < 10)
        std::printf("Fail at row %d: (gold) %f != %f (computed)\n", i, gold[i], src[i]);
      ++num_errors;
    }
  }
  return num_errors;
}

// .vec: 공백 구분 텍스트 float 벡터
static bool read_vec_text(const std::string& path, std::vector<TYPE>& x, int expectedN){
  std::ifstream ifs(path);
  if (!ifs) return false;
  x.clear(); x.reserve(expectedN);
  double v;
  while (ifs >> v){
    x.push_back((TYPE)v);
  }
  return (int)x.size() == expectedN;
}

// ------------------------ Matrix Market -> COO ------------------------
struct COO {
  int rows=0, cols=0;
  std::vector<int>   I, J;
  std::vector<TYPE>  V;
};

static bool read_matrix_market(const std::string& path, COO& coo){
  std::ifstream ifs(path);
  if (!ifs){
    std::fprintf(stderr, "Cannot open %s\n", path.c_str());
    return false;
  }
  std::string line;
  if (!std::getline(ifs, line)) return false;
  if (line.rfind("%%MatrixMarket", 0) != 0){
    std::fprintf(stderr, "Not a MatrixMarket file\n");
    return false;
  }
  // 스펙 간략 대응: coordinate real general
  // 주석/공백 스킵 후 크기/NNZ 라인
  while (std::getline(ifs, line)){
    if (line.empty() || line[0]=='%') continue;
    std::istringstream iss(line);
    int M,N,NNZ; if (!(iss>>M>>N>>NNZ)){ std::fprintf(stderr,"Bad size line\n"); return false; }
    coo.rows=M; coo.cols=N; coo.I.reserve(NNZ); coo.J.reserve(NNZ); coo.V.reserve(NNZ);
    for (int k=0;k<NNZ;++k){
      if (!std::getline(ifs, line)) return false;
      if (line.empty() || line[0]=='%'){ --k; continue; }
      std::istringstream t(line);
      int i,j; double v;
      if (!(t >> i >> j >> v)) return false;
      coo.I.push_back(i-1);
      coo.J.push_back(j-1);
      coo.V.push_back((TYPE)v);
    }
    break;
  }
  return true;
}

// ------------------------ COO -> CSR ------------------------
struct CSR {
  int rows=0, cols=0;
  std::vector<int>   Ap; // size rows+1
  std::vector<int>   Aj; // size nnz
  std::vector<TYPE>  Ax; // size nnz
};

static void coo_to_csr(const COO& coo, CSR& csr){
  csr.rows = coo.rows; csr.cols = coo.cols;
  int M = csr.rows; size_t NNZ = coo.I.size();
  csr.Ap.assign(M+1, 0);
  csr.Aj.resize(NNZ);
  csr.Ax.resize(NNZ);

  // row counts
  for (size_t k=0;k<NNZ;++k) ++csr.Ap[coo.I[k]];
  // prefix
  int sum=0;
  for (int r=0;r<M;++r){ int c=csr.Ap[r]; csr.Ap[r]=sum; sum+=c; }
  csr.Ap[M]=sum;

  // scatter (stable)
  std::vector<int> next(csr.Ap.begin(), csr.Ap.end());
  for (size_t k=0;k<NNZ;++k){
    int r = coo.I[k];
    int p = next[r]++;
    csr.Aj[p] = coo.J[k];
    csr.Ax[p] = coo.V[k];
  }

  // 각 행 안에서 column 정렬(선택) — 안정성을 위해
  for (int r=0;r<M;++r){
    int s=csr.Ap[r], e=csr.Ap[r+1], len=e-s;
    std::vector<int> idx(len);
    for (int t=0;t<len;++t) idx[t]=s+t;
    std::sort(idx.begin(), idx.end(), [&](int a,int b){ return csr.Aj[a]<csr.Aj[b]; });
    std::vector<int>   newJ; newJ.reserve(len);
    std::vector<TYPE>  newV; newV.reserve(len);
    for (int t=0;t<len;++t){ newJ.push_back(csr.Aj[idx[t]]); newV.push_back(csr.Ax[idx[t]]); }
    for (int t=0;t<len;++t){ csr.Aj[s+t]=newJ[t]; csr.Ax[s+t]=newV[t]; }
  }
}

// ------------------------ CSR -> JDS ------------------------
struct JDS {
  // JDS 배열(Parboil 스타일)
  // data/indices : 대각(diagonal)별 연속 저장, 각 대각 시작은 jds_ptr[k]
  // perm        : 행 재배열(원래 행 idx)
  // sh_zcnt     : 블록별 유효 대각 개수 (크기 = numBlocks = ceil(dim/pad))
  // depth       : 대각 수 = max row nnz
  // len         : data/indices 길이(=nnz after sort)
  int dim=0, depth=0, len=0, nzcnt_len=0; // nzcnt_len = numBlocks
  std::vector<TYPE> data;
  std::vector<int>  indices;
  std::vector<int>  perm;
  std::vector<int>  jds_ptr;   // size = depth
  std::vector<int>  sh_zcnt;   // size = nzcnt_len
};

static void csr_to_jds(const CSR& csr, int pad, JDS& jds){
  const int M = csr.rows;
  jds.dim = M;
  // 행별 nnz
  std::vector<int> row_nnz(M);
  int depth = 0;
  for (int r=0;r<M;++r){
    row_nnz[r] = csr.Ap[r+1]-csr.Ap[r];
    depth = std::max(depth, row_nnz[r]);
  }
  jds.depth = depth;

  // perm: nnz 내림차순으로 행 인덱스 정렬
  jds.perm.resize(M);
  for (int r=0;r<M;++r) jds.perm[r]=r;
  std::stable_sort(jds.perm.begin(), jds.perm.end(),
                   [&](int a,int b){ return row_nnz[a] > row_nnz[b]; });

  // 각 대각의 길이: diag_len[k] = nnz가 k+1 이상인 행의 수
  std::vector<int> diag_len(depth, 0);
  for (int k=0;k<depth;++k){
    int cnt=0;
    for (int p=0;p<M;++p){
      int r = jds.perm[p];
      if (row_nnz[r] > k) ++cnt;
      else break; // 정렬된 상태라 break 가능
    }
    diag_len[k]=cnt;
  }

  // jds_ptr = 각 대각의 시작 오프셋 (누적합)
  jds.jds_ptr.resize(depth);
  int off=0;
  for (int k=0;k<depth;++k){ jds.jds_ptr[k]=off; off += diag_len[k]; }
  jds.len = off;

  jds.data.assign(jds.len, (TYPE)0);
  jds.indices.assign(jds.len, 0);

  // 각 행의 k번째(0-based) 비제로를 찾아 대각 k의 [jds_ptr[k] + p] 위치에 기록
  // CSR 안에서 행의 비제로들은 이미 열 인덱스 오름차순
  // p: permuted row order index(=thread 내 행 순번)
  for (int p=0; p<M; ++p){
    int r = jds.perm[p];
    int s = csr.Ap[r], e = csr.Ap[r+1];
    int nnz = e - s;
    for (int k=0;k<nnz;++k){
      int j = jds.jds_ptr[k] + p; // 해당 대각의 p번째 원소
      jds.data[j]   = csr.Ax[s+k];
      jds.indices[j]= csr.Aj[s+k];
    }
  }

  // sh_zcnt: 블록(=warp)별 유효 대각 수
  int numBlocks = (M + pad - 1) / pad;
  jds.nzcnt_len = numBlocks;
  jds.sh_zcnt.assign(numBlocks, 0);
  for (int b=0;b<numBlocks;++b){
    int row_start = b*pad;
    // 각 대각에서 row_start < diag_len[k] 인 k의 개수
    int bound=0;
    for (int k=0;k<depth;++k){
      if (row_start < diag_len[k]) ++bound; else break;
    }
    jds.sh_zcnt[b]=bound;
  }
}

// ------------------------ CPU 참조 (검증용) ------------------------
static void spmv_jds_cpu(const JDS& jds, const std::vector<TYPE>& x, std::vector<TYPE>& y){
  y.assign(jds.dim, (TYPE)0);
  // 원본 Parboil 커널과 동일한 수식:
  // 각 permuted row i에 대해 sum_k data[jds_ptr[k]+i] * x[indices[jds_ptr[k]+i]]
  // 결과는 원래 행 위치 d_perm[i]에 저장
  for (int i=0;i<jds.dim;++i){
    TYPE sum=0;
    int bound = jds.depth; // 실제로는 유효 대각까지만 보지만, out-of-range는 diag_len으로 비어있음
    for (int k=0;k<bound;++k){
      int j = jds.jds_ptr[k] + i;
      if (j >= jds.len) break; // 안전 가드
      // 유효한지 확인: i가 이 대각의 길이보다 작은지? (j 인덱스가 data 범위 내인지로 충분)
      sum += jds.data[j] * x[jds.indices[j]];
    }
    // perm 역매핑: perm[p]=원행, 여기선 p=i의 원행
    y[jds.perm[i]] = sum;
  }
}

// ------------------------ CUDA 커널 ------------------------
__global__ void spmv_jds_naive_kernel(
    float* __restrict__ d_Ax,        // dim
    const float* __restrict__ d_data,// len
    const int*   __restrict__ d_idx, // len
    const int*   __restrict__ d_perm,// dim
    const float* __restrict__ d_x,   // cols
    int dim,
    const int*   __restrict__ jds_ptr,   // depth
    const int*   __restrict__ sh_zcnt)   // nzcnt_len == numBlocks
{
  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int i  = bx * blockDim.x + tx;  // permuted row
  if (i >= dim) return;

  float sum = 0.0f;
  int bound = sh_zcnt[bx];        // 이 블록에서 유효한 대각 개수
  for (int k=0;k<bound;++k){
    int j = jds_ptr[k] + i;
    sum += d_data[j] * d_x[d_idx[j]];
  }
  d_Ax[d_perm[i]] = sum;
}

// ------------------------ main ------------------------
int main(int argc, char** argv){
  std::string in_pair; // "matrix.mtx,vector.vec"
  std::string out_path;
  int pad = 32;

  int opt;
  while ((opt = getopt(argc, argv, "i:o:p:h")) != -1){
    switch(opt){
      case 'i': in_pair = optarg; break;
      case 'o': out_path = optarg; break;
      case 'p': pad = std::max(1, std::atoi(optarg)); break;
      case 'h': usage(argv[0]); return 0;
      default : usage(argv[0]); return 1;
    }
  }
  if (in_pair.empty()){
    usage(argv[0]); return 1;
  }

  // "mtx,vec" 파싱
  std::string mtx_path, vec_path;
  {
    auto pos = in_pair.find(',');
    if (pos == std::string::npos){
      std::fprintf(stderr, "Expecting -i <matrix.mtx,vector.vec>\n");
      return 1;
    }
    mtx_path = in_pair.substr(0,pos);
    vec_path = in_pair.substr(pos+1);
  }

  // Matrix Market 로드 -> COO -> CSR -> JDS
  COO coo;
  if (!read_matrix_market(mtx_path, coo)) return 1;
  CSR csr; coo_to_csr(coo, csr);
  if (csr.rows != csr.cols){
    std::fprintf(stderr,"Warning: non-square matrix %dx%d; continuing.\n", csr.rows, csr.cols);
  }
  JDS jds; csr_to_jds(csr, pad, jds);

  // x 벡터 로드 (크기 csr.cols)
  std::vector<TYPE> h_x(csr.cols);
  if (!read_vec_text(vec_path, h_x, csr.cols)){
    std::fprintf(stderr, "Failed to read vector '%s' with length %d\n", vec_path.c_str(), csr.cols);
    return 1;
  }

  // 디바이스 버퍼
  float *d_Ax=nullptr, *d_data=nullptr, *d_x=nullptr;
  int   *d_idx=nullptr, *d_perm=nullptr, *d_jds_ptr=nullptr, *d_shzcnt=nullptr;

  CUDA_CHECK(hipMalloc(&d_Ax,      sizeof(float)*jds.dim));
  CUDA_CHECK(hipMalloc(&d_data,    sizeof(float)*jds.len));
  CUDA_CHECK(hipMalloc(&d_idx,     sizeof(int)  *jds.len));
  CUDA_CHECK(hipMalloc(&d_perm,    sizeof(int)  *jds.dim));
  CUDA_CHECK(hipMalloc(&d_x,       sizeof(float)*csr.cols));
  CUDA_CHECK(hipMalloc(&d_jds_ptr, sizeof(int)  *jds.depth));
  CUDA_CHECK(hipMalloc(&d_shzcnt,  sizeof(int)  *jds.nzcnt_len));

  CUDA_CHECK(hipMemcpy(d_data,    jds.data.data(),   sizeof(float)*jds.len,      hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_idx,     jds.indices.data(),sizeof(int)  *jds.len,      hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_perm,    jds.perm.data(),   sizeof(int)  *jds.dim,      hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_x,       h_x.data(),        sizeof(float)*csr.cols,     hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_jds_ptr, jds.jds_ptr.data(),sizeof(int)  *jds.depth,    hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_shzcnt,  jds.sh_zcnt.data(),sizeof(int)  *jds.nzcnt_len,hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemset(d_Ax, 0, sizeof(float)*jds.dim));

  // 런치 구성: 원본과 동일 개념 — blockDim.x = pad, gridDim.x = ceil(dim/pad)
  dim3 block(pad, 1, 1);
  dim3 grid((jds.dim + pad - 1) / pad, 1, 1);

  std::printf("grid=%d, block=%d, dim=%d, depth=%d, len=%d\n",
              grid.x, block.x, jds.dim, jds.depth, jds.len);

  auto t0 = std::chrono::high_resolution_clock::now();
  spmv_jds_naive_kernel<<<grid, block>>>(d_Ax, d_data, d_idx, d_perm, d_x,
                                         jds.dim, d_jds_ptr, d_shzcnt);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
  auto t1 = std::chrono::high_resolution_clock::now();
  double ms = std::chrono::duration<double, std::milli>(t1 - t0).count();
  std::printf("Elapsed: %.3f ms\n", ms);

  // 결과 수거
  std::vector<float> h_Ax(jds.dim);
  CUDA_CHECK(hipMemcpy(h_Ax.data(), d_Ax, sizeof(float)*jds.dim, hipMemcpyDeviceToHost));

  // out.bin 요청시 저장 (Parboil 포맷과 동일: int N + float[N])
  if (!out_path.empty()){
    if (write_bin_with_size(out_path.c_str(), h_Ax.data(), jds.dim) != 0){
      std::fprintf(stderr, "Failed to write %s\n", out_path.c_str());
    } else {
      std::printf("Wrote %s\n", out_path.c_str());
    }
  }

  // gold 비교 (matrix.mtx -> matrix.gold)
  {
    std::string gold_path = replace_ext(mtx_path, ".gold");
    int gN=0; float* gbuf = read_gold_file(gold_path.c_str(), &gN);
    if (gbuf){
      if (gN != jds.dim){
        std::printf("error: gold data size mismatch: current=%d, expected=%d\n", jds.dim, gN);
      } else {
        int errors = compare_floats(h_Ax.data(), gbuf, gN);
        if (errors > 0) std::printf("FAILED!\n"); else std::printf("PASSED!\n");
      }
      std::free(gbuf);
    } else {
      std::printf("(gold '%s' not found — skipping check)\n", gold_path.c_str());
    }
  }

  // 정리
  hipFree(d_Ax); hipFree(d_data); hipFree(d_idx);
  hipFree(d_perm); hipFree(d_x);
  hipFree(d_jds_ptr); hipFree(d_shzcnt);
  return 0;
}
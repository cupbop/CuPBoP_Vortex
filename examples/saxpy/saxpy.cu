// saxpy_cuda.cu
// CUDA version of the provided OpenCL SAXPY sample (1:1 behavior)
// Build: nvcc -O3 -std=c++17 -o saxpy_cuda saxpy_cuda.cu
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <unistd.h>
#include <chrono>
#include <vector>
#include <iostream>

#ifndef BLOCK_SIZE
// Match the OpenCL sample's local_work_size = 1 for strict 1:1 behavior.
// You can -DBLOCK_SIZE=256 for faster runs if desired.
#define BLOCK_SIZE 512
#endif

#define CUDA_CHECK(expr)                                                     \
  do {                                                                       \
    hipError_t _err = (expr);                                               \
    if (_err == hipSuccess) break;                                          \
    fprintf(stderr, "CUDA Error: %s returned %d (%s) at %s:%d\n",            \
            #expr, int(_err), hipGetErrorString(_err), __FILE__, __LINE__); \
    std::abort();                                                            \
  } while (0)

#define CUDA_CHECK_LAST()                                                    \
  do {                                                                       \
    hipError_t _err = hipGetLastError();                                   \
    if (_err == hipSuccess) break;                                          \
    fprintf(stderr, "CUDA Kernel Error: %s at %s:%d\n",                      \
            hipGetErrorString(_err), __FILE__, __LINE__);                   \
    std::abort();                                                            \
  } while (0)

static bool almost_equal(float a, float b, int ulp = 4) {
  union fi_t { int i; float f; };
  fi_t fa, fb;
  fa.f = a;
  fb.f = b;
  return std::abs(fa.i - fb.i) <= ulp;
}

__global__ void saxpy_kernel(const float* __restrict__ src,
                             float* __restrict__ dst,
                             float factor, int n)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    dst[i] += src[i] * factor;
  }
}

static void show_usage() {
  std::printf("Usage: [-n size] [-h: help]\n");
}

int main(int argc, char** argv) {
  int size = 1024;

  // parse args (same flags as OpenCL sample)
  int c;
  while ((c = getopt(argc, argv, "n:h")) != -1) {
    switch (c) {
      case 'n': size = std::atoi(optarg); break;
      case 'h': show_usage(); return 0;
      default: show_usage(); return -1;
    }
  }
  std::printf("Workload size=%d\n", size);

  // Print device info (rough equivalent to clGetDeviceInfo)
  int dev = 0;
  CUDA_CHECK(hipGetDevice(&dev));
  hipDeviceProp_t prop{};
  CUDA_CHECK(hipGetDeviceProperties(&prop, dev));
  std::printf("Using device: %s (SM %d.%d)\n", prop.name, prop.major, prop.minor);

  size_t nbytes = sizeof(float) * size;

  // Host buffers (same init flow as OpenCL sample)
  std::vector<float> src_vec(size), dst_vec(size, 0.0f), ref_vec(size, 0.0f);
  float factor = (float(rand()) / float(RAND_MAX)) * 100.0f;
  for (int i = 0; i < size; ++i) {
    src_vec[i] = (float(rand()) / float(RAND_MAX)) * 100.0f;
  }

  // Reference computation: identical math as the OpenCL host-side loop with local_work_size=1
  for (int i = 0; i < size; ++i) {
    ref_vec[i] += src_vec[i] * factor;
  }

  // Device buffers (equivalent to clCreateBuffer + clEnqueueWriteBuffer)
  float *d_src = nullptr, *d_dst = nullptr;
  CUDA_CHECK(hipMalloc(&d_src, nbytes));
  CUDA_CHECK(hipMalloc(&d_dst, nbytes));
  CUDA_CHECK(hipMemcpy(d_src, src_vec.data(), nbytes, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_dst, dst_vec.data(), nbytes, hipMemcpyHostToDevice));

  // Launch (equivalent to clEnqueueNDRangeKernel with global=size and local=1 by default)
  dim3 block(BLOCK_SIZE);
  dim3 grid((size + block.x - 1) / block.x);

  std::printf("enqueue kernel\n");
  std::printf("global_work_size=%d\n", size);
  auto t0 = std::chrono::high_resolution_clock::now();
  saxpy_kernel<<<grid, block>>>(d_src, d_dst, factor, size);
  CUDA_CHECK_LAST();
  CUDA_CHECK(hipDeviceSynchronize());
  auto t1 = std::chrono::high_resolution_clock::now();
  double elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(t1 - t0).count();
  std::printf("Elapsed time: %lg ms\n", elapsed);

  // Read back and verify (equivalent to clEnqueueReadBuffer + comparison)
  CUDA_CHECK(hipMemcpy(dst_vec.data(), d_dst, nbytes, hipMemcpyDeviceToHost));

  int errors = 0;
  for (int i = 0; i < size; ++i) {
    if (!almost_equal(dst_vec[i], ref_vec[i])) {
      if (errors < 100) {
        std::printf("*** error: [%d] expected=%f, actual=%f, factor=%f\n",
                    i, ref_vec[i], dst_vec[i], factor);
      }
      ++errors;
    }
  }
  if (errors == 0) std::puts("PASSED!");
  else std::printf("FAILED! - %d errors\n", errors);

  // Cleanup
  CUDA_CHECK(hipFree(d_src));
  CUDA_CHECK(hipFree(d_dst));
  return errors;
}

#include <hip/hip_runtime.h>
#include <stdio.h>

#ifdef RD_WG_SIZE_0_0
        #define BLOCK_SIZE RD_WG_SIZE_0_0
#elif defined(RD_WG_SIZE_0)
        #define BLOCK_SIZE RD_WG_SIZE_0
#elif defined(RD_WG_SIZE)
        #define BLOCK_SIZE RD_WG_SIZE
#else
        #define BLOCK_SIZE 16
#endif


__global__ void 
lud_diagonal(float *m, int matrix_dim, int offset)
{
  int i,j;
  __shared__ float shadow[BLOCK_SIZE][BLOCK_SIZE];

  int array_offset = offset*matrix_dim+offset;
  for(i=0; i < BLOCK_SIZE; i++){
    if (array_offset+threadIdx.x < matrix_dim*matrix_dim)
      shadow[i][threadIdx.x]=m[array_offset+threadIdx.x];
    array_offset += matrix_dim;
  }
  __syncthreads();
  for(i=0; i < BLOCK_SIZE-1; i++) {

    if (threadIdx.x>i){
      for(j=0; j < i; j++)
        shadow[threadIdx.x][i] -= shadow[threadIdx.x][j]*shadow[j][i];
      shadow[threadIdx.x][i] /= shadow[i][i];
    }

    __syncthreads();
    if (threadIdx.x>i){

      for(j=0; j < i+1; j++)
        shadow[i+1][threadIdx.x] -= shadow[i+1][j]*shadow[j][threadIdx.x];
    }
    __syncthreads();
  }

  /* 
     The first row is not modified, it
     is no need to write it back to the
     global memory

   */
  array_offset = (offset+1)*matrix_dim+offset;
  for(i=1; i < BLOCK_SIZE; i++){
    if (array_offset+threadIdx.x < matrix_dim*matrix_dim)
      m[array_offset+threadIdx.x]=shadow[i][threadIdx.x];
    array_offset += matrix_dim;
  }
}

__global__ void
lud_perimeter(float *m, int matrix_dim, int offset)
{
  __shared__ float dia[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float peri_row[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float peri_col[BLOCK_SIZE][BLOCK_SIZE];

  int i,j, array_offset;
  int idx;

  if (threadIdx.x < BLOCK_SIZE) {
    idx = threadIdx.x;
    
    array_offset = offset*matrix_dim+offset;
    for (i=0; i < BLOCK_SIZE/2; i++){
      if (array_offset+idx < matrix_dim*matrix_dim)
        dia[i][idx]=m[array_offset+idx];
      array_offset += matrix_dim;
    }
    
    array_offset = offset*matrix_dim+offset;
    for (i=0; i < BLOCK_SIZE; i++) {
      if(array_offset+(blockIdx.x+1)*BLOCK_SIZE+idx < matrix_dim*matrix_dim)
        peri_row[i][idx]=m[array_offset+(blockIdx.x+1)*BLOCK_SIZE+idx];
      array_offset += matrix_dim;
    }

  } else {
    idx = threadIdx.x-BLOCK_SIZE;
    
    array_offset = (offset+BLOCK_SIZE/2)*matrix_dim+offset;
    for (i=BLOCK_SIZE/2; i < BLOCK_SIZE; i++){
      if(array_offset+idx < matrix_dim*matrix_dim)
        dia[i][idx]=m[array_offset+idx];
      array_offset += matrix_dim;
    }
    
    array_offset = (offset+(blockIdx.x+1)*BLOCK_SIZE)*matrix_dim+offset;
    for (i=0; i < BLOCK_SIZE; i++) {
      if(array_offset+idx < matrix_dim*matrix_dim)
        peri_col[i][idx] = m[array_offset+idx];
      array_offset += matrix_dim;
    }
  
  }
  __syncthreads();

/* this version works ok on hardware, but not gpgpusim
 **************************************************************
  if (threadIdx.x < BLOCK_SIZE) { //peri-row
    idx=threadIdx.x;
    for(i=1; i < BLOCK_SIZE; i++){
      for (j=0; j < i; j++)
        peri_row[i][idx]-=dia[i][j]*peri_row[j][idx];
    }

    
    array_offset = (offset+1)*matrix_dim+offset;
    for(i=1; i < BLOCK_SIZE; i++){
      m[array_offset+(blockIdx.x+1)*BLOCK_SIZE+idx] = peri_row[i][idx];
      array_offset += matrix_dim;
    }
  } else { //peri-col
    idx=threadIdx.x - BLOCK_SIZE;
    for(i=0; i < BLOCK_SIZE; i++){
      for(j=0; j < i; j++)
        peri_col[idx][i]-=peri_col[idx][j]*dia[j][i];
      peri_col[idx][i] /= dia[i][i];
    }

    __syncthreads();
    
    array_offset = (offset+(blockIdx.x+1)*BLOCK_SIZE)*matrix_dim+offset;
    for(i=0; i < BLOCK_SIZE; i++){
      m[array_offset+idx] =  peri_col[i][idx];
      array_offset += matrix_dim;
    }
  }
***************************************************************/
  if (threadIdx.x < BLOCK_SIZE) { //peri-row
    idx=threadIdx.x;
    for(i=1; i < BLOCK_SIZE; i++){
      for (j=0; j < i; j++)
        peri_row[i][idx]-=dia[i][j]*peri_row[j][idx];
    }
  } else { //peri-col
    idx=threadIdx.x - BLOCK_SIZE;
    for(i=0; i < BLOCK_SIZE; i++){
      for(j=0; j < i; j++)
        peri_col[idx][i]-=peri_col[idx][j]*dia[j][i];
      peri_col[idx][i] /= dia[i][i];
    }
  }

  __syncthreads();
    
  if (threadIdx.x < BLOCK_SIZE) { //peri-row
    idx=threadIdx.x;
    array_offset = (offset+1)*matrix_dim+offset;
    for(i=1; i < BLOCK_SIZE; i++){
      if(array_offset+(blockIdx.x+1)*BLOCK_SIZE+idx < matrix_dim*matrix_dim)
        m[array_offset+(blockIdx.x+1)*BLOCK_SIZE+idx] = peri_row[i][idx];
      array_offset += matrix_dim;
    }
  } else { //peri-col
    idx=threadIdx.x - BLOCK_SIZE;
    array_offset = (offset+(blockIdx.x+1)*BLOCK_SIZE)*matrix_dim+offset;
    for(i=0; i < BLOCK_SIZE; i++){
      if(array_offset+idx < matrix_dim*matrix_dim)
        m[array_offset+idx] =  peri_col[i][idx];
      array_offset += matrix_dim;
    }
  }

}

__global__ void
lud_internal(float *m, const int matrix_dim, const int offset)
{
  __shared__ float peri_row[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float peri_col[BLOCK_SIZE][BLOCK_SIZE];



  for (int i=0; i < BLOCK_SIZE; i++)
  {
    for (int j=0; j < BLOCK_SIZE; j++)
      {
        peri_row[i][j] = 0.0f;
        peri_col[i][j] = 0.0f;
      }
  }

  __syncthreads();

  int i;
  float sum;
  //int bx = blockIdx.x;
  //int by = blockIdx.y;

  int global_row_id = offset + (blockIdx.x+1)*BLOCK_SIZE;
  int global_col_id = offset + (blockIdx.y+1)*BLOCK_SIZE;

  if (((offset+threadIdx.y)*matrix_dim+global_col_id+threadIdx.x) < matrix_dim*matrix_dim)
    peri_row[threadIdx.y][threadIdx.x] = m[(offset+threadIdx.y)*matrix_dim+global_col_id+threadIdx.x];
  if (((global_row_id+threadIdx.y)*matrix_dim+offset+threadIdx.x) < matrix_dim*matrix_dim)
    peri_col[threadIdx.y][threadIdx.x] = m[(global_row_id+threadIdx.y)*matrix_dim+offset+threadIdx.x];

  __syncthreads();
  //print peri_col if threadidx.x=0 and threadidx.y=0
  /*
  if (threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x==0 && blockIdx.y==0)
  {
    printf("peri_col\n");
    for (int i=0; i < BLOCK_SIZE; i++)
    {
      
      for (int j=0; j < BLOCK_SIZE; j++)
      {
      printf("%.2f ", peri_col[i][j]);
      }
      printf("\n");
    }
    printf("peri_row\n");
      for (int i=0; i < BLOCK_SIZE; i++)
      {
        for (int j=0; j < BLOCK_SIZE; j++)
        {
        printf("%.2f ", peri_row[i][j]);
        }
        printf("\n");
      }
    }
  */
      


  //if (threadIdx.x > BLOCK_SIZE || threadIdx.y > BLOCK_SIZE || threadIdx.x < 0 || threadIdx.y < 0 )
  //  printf("threadIdx.x=%d, threadIdx.y=%d, global_row_id=%d, global_col_id=%d\n", threadIdx.x, threadIdx.y, global_row_id, global_col_id);
  sum = 0.0f;
  i = 0;
  for (i=0; i < BLOCK_SIZE; i++)
  {
      if(peri_col[threadIdx.y][i] > 0.001 && peri_row[i][threadIdx.x] > 0.001)
      {
      sum += peri_col[threadIdx.y][i] * peri_row[i][threadIdx.x];
      //printf("sum+=%f %f\n", peri_col[threadIdx.y][i], peri_row[i][threadIdx.x]);
      }
  }
      int col_idx = ((global_row_id+threadIdx.y)*matrix_dim+global_col_id+threadIdx.x)/64;
      int row_idx = ((global_row_id+threadIdx.y)*matrix_dim+global_col_id+threadIdx.x)%64;
  //printf("accessing index: %d %d\n", col_idx, row_idx);
  if (((global_row_id+threadIdx.y)*matrix_dim+global_col_id+threadIdx.x) < matrix_dim*matrix_dim)
  {
    m[(global_row_id+threadIdx.y)*matrix_dim+global_col_id+threadIdx.x] -= sum;
    //printf("m[%d] -= %f\n", (global_row_id+threadIdx.y)*matrix_dim+global_col_id+threadIdx.x, sum);
    //printf("matrix_dim=%d global_row_id=%d global_col_id=%d\n", matrix_dim, global_row_id, global_col_id);

  }


}


void lud_cuda(float *m, int matrix_dim)
{
  int i=0;
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  //float *m_debug = (float*)malloc(matrix_dim*matrix_dim*sizeof(float));

  //
  for (i=0; i < matrix_dim-BLOCK_SIZE; i += BLOCK_SIZE) {
  //i=0;
      lud_diagonal<<<1, BLOCK_SIZE>>>(m, matrix_dim, i);
      lud_perimeter<<<(matrix_dim-i)/BLOCK_SIZE-1, BLOCK_SIZE*2>>>(m, matrix_dim, i);
      dim3 dimGrid((matrix_dim-i)/BLOCK_SIZE-1, (matrix_dim-i)/BLOCK_SIZE-1);
      lud_internal<<<dimGrid, dimBlock>>>(m, matrix_dim, i); 
  }
  lud_diagonal<<<1,BLOCK_SIZE>>>(m, matrix_dim, i);
}
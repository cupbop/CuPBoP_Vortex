// sgemm_cuda.cu - CUDA counterpart of the provided OpenCL SGEMM sample (block size tunable)
// Build: nvcc -O3 -std=c++17 -o sgemm_cuda sgemm_cuda.cu
// Run  : ./sgemm_cuda -n 32 -b16x16   (or -bx 16 -by 16)

#include <cstdio>
#include <cstdlib>
#include <cassert>
#include <cstdint>
#include <cstring>
#include <unistd.h>
#include <chrono>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>

#ifndef TYPE
#define TYPE float
#endif

#define FLOAT_ULP 6

// CUDA error check macro
#define CUDA_CHECK(_expr)                                                     \
  do {                                                                        \
    hipError_t _err = (_expr);                                               \
    if (_err == hipSuccess) break;                                           \
    fprintf(stderr, "CUDA Error: '%s' returned %d (%s)!\n",                   \
            #_expr, (int)_err, hipGetErrorString(_err));                     \
    cleanup();                                                                \
    exit(-1);                                                                 \
  } while (0)

// ---------- Comparator ----------
template <typename T>
class Comparator {};

template <>
class Comparator<int> {
public:
  static const char* type_str() { return "integer"; }
  static int generate() { return rand(); }
  static bool compare(int a, int b, int index, int errors) {
    if (a != b) {
      if (errors < 100) {
        printf("*** error: [%d] expected=%d, actual=%d\n", index, a, b);
      }
      return false;
    }
    return true;
  }
};

template <>
class Comparator<float> {
public:
  static const char* type_str() { return "float"; }
  static float generate() { return static_cast<float>(rand()) / RAND_MAX; }
  static bool compare(float a, float b, int index, int errors) {
    union fi_t { float f; int32_t i; };
    fi_t fa, fb;
    fa.f = a; fb.f = b;
    auto d = std::abs(fa.i - fb.i);
    if (d > FLOAT_ULP) {
      if (errors < 100) {
        printf("*** error: [%d] expected=%f, actual=%f\n", index, a, b);
      }
      return false;
    }
    return true;
  }
};

// ---------- CPU reference (column-major, identical indexing) ----------
static void sgemm_cpu(TYPE* C, const TYPE* A, const TYPE* B, int M, int N, int K) {
  for (int m = 0; m < M; ++m) {
    for (int n = 0; n < N; ++n) {
      TYPE acc = 0;
      for (int k = 0; k < K; ++k) {
        acc += A[k * M + m] * B[n * K + k]; // A(r,k) * B(k,n) in column-major
      }
      C[n * M + m] = acc; // C(r,n) in column-major
    }
  }
}

// ---------- CUDA kernel ----------
template <typename T>
__global__ void sgemm_kernel(const T* __restrict__ A,
                             const T* __restrict__ B,
                             T* __restrict__ C,
                             int N)
{
  const int r = blockIdx.x * blockDim.x + threadIdx.x; // row id
  const int c = blockIdx.y * blockDim.y + threadIdx.y; // col id
  if (r >= N || c >= N) return;

  T acc = 0;
  for (int k = 0; k < N; ++k) {
    acc += A[k * N + r] * B[c * N + k];
  }
  C[c * N + r] = acc;
}

// ---------- Globals for cleanup ----------
static TYPE* d_A = nullptr;
static TYPE* d_B = nullptr;
static TYPE* d_C = nullptr;

static void cleanup() {
  if (d_A) { hipFree(d_A); d_A = nullptr; }
  if (d_B) { hipFree(d_B); d_B = nullptr; }
  if (d_C) { hipFree(d_C); d_C = nullptr; }
}

// ---------- CLI ----------
static void show_usage() {
  printf("Usage: sgemm_cuda [-n size] [-b WxH | -bx W -by H] [-h]\n");
  printf("  -n   : matrix size (N x N), N >= 2 (default 32)\n");
  printf("  -b   : block size in 'WxH' format (e.g., -b32x8)\n");
  printf("  -bx  : blockDim.x (e.g., -bx 32)\n");
  printf("  -by  : blockDim.y (e.g., -by 8)\n");
  printf("  -h   : help\n");
}

static int sizeN = 32;
static int block_x = 16;
static int block_y = 16;

static void parse_b_arg(const char* s) {
  int w = 0, h = 0;
  if (std::sscanf(s, "%dx%d", &w, &h) == 2 && w > 0 && h > 0) {
    block_x = w; block_y = h;
  } else {
    fprintf(stderr, "Invalid -b format. Use -b32x8\n");
    std::exit(-1);
  }
}

static void parse_args(int argc, char** argv) {
  int c;
  while ((c = getopt(argc, argv, "n:b:h")) != -1) {
    switch (c) {
      case 'n': sizeN = std::atoi(optarg); break;
      case 'b': parse_b_arg(optarg); break;
      case 'h': show_usage(); std::exit(0);
      default : show_usage(); std::exit(-1);
    }
  }
  // Optional -bx / -by support
  for (int i = 1; i < argc; ++i) {
    if (std::strcmp(argv[i], "-bx") == 0 && i + 1 < argc) block_x = std::atoi(argv[i + 1]);
    if (std::strcmp(argv[i], "-by") == 0 && i + 1 < argc) block_y = std::atoi(argv[i + 1]);
  }
  if (sizeN < 2) {
    fprintf(stderr, "Error: invalid size!\n");
    std::exit(-1);
  }
  if (block_x <= 0 || block_y <= 0) {
    fprintf(stderr, "Error: invalid block size (%d x %d)\n", block_x, block_y);
    std::exit(-1);
  }
  printf("Workload size=%d, block=%dx%d\n", sizeN, block_x, block_y);
}

// ---------- main ----------
int main(int argc, char** argv) {
  parse_args(argc, argv);

  const int N = sizeN;
  const size_t size_sq = static_cast<size_t>(N) * N;
  const size_t nbytes = size_sq * sizeof(TYPE);

  // Host buffers
  std::vector<TYPE> h_A(size_sq);
  std::vector<TYPE> h_B(size_sq);
  std::vector<TYPE> h_C(size_sq, 0);
  std::vector<TYPE> h_ref(size_sq, 0);

  // Init inputs
  for (size_t i = 0; i < size_sq; ++i) {
    h_A[i] = Comparator<TYPE>::generate();
    h_B[i] = Comparator<TYPE>::generate();
  }

  // Device buffers
  CUDA_CHECK(hipMalloc(&d_A, nbytes));
  CUDA_CHECK(hipMalloc(&d_B, nbytes));
  CUDA_CHECK(hipMalloc(&d_C, nbytes));

  // Upload
  CUDA_CHECK(hipMemcpy(d_A, h_A.data(), nbytes, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_B, h_B.data(), nbytes, hipMemcpyHostToDevice));

  // Launch config (tunable)
  const dim3 block(block_x, block_y, 1);
  const dim3 grid((N + block_x - 1) / block_x,
                  (N + block_y - 1) / block_y,
                  1);

  auto t0 = std::chrono::high_resolution_clock::now();
  sgemm_kernel<TYPE><<<grid, block>>>(d_A, d_B, d_C, N);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
  auto t1 = std::chrono::high_resolution_clock::now();
  double elapsed_ms = std::chrono::duration_cast<std::chrono::milliseconds>(t1 - t0).count();
  printf("Elapsed time: %lg ms (grid=%dx%d, block=%dx%d)\n",
         elapsed_ms, grid.x, grid.y, block.x, block.y);

  // Download
  CUDA_CHECK(hipMemcpy(h_C.data(), d_C, nbytes, hipMemcpyDeviceToHost));

  // Reference & verify
  sgemm_cpu(h_ref.data(), h_A.data(), h_B.data(), N, N, N);
  int errors = 0;
  for (size_t i = 0; i < size_sq; ++i) {
    if (!Comparator<TYPE>::compare(h_C[i], h_ref[i], static_cast<int>(i), errors)) {
      ++errors;
    }
  }
  if (errors) {
    printf("FAILED! - %d errors\n", errors);
  } else {
    printf("PASSED!\n");
  }

  cleanup();
  return errors;
}
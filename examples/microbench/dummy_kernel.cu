
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void saxpy(void)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  printf("block_id:%d thread_id:%d \n", i);
}

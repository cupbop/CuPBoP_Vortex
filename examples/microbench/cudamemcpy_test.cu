
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(void)
{
  int N = 1<<20;
  float *x, *y, *rx, *ry, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));
  rx = (float*)malloc(N*sizeof(float));
  ry = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  // saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

  hipMemcpy(rx, d_x, N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(ry, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  float sumError = 0.0f;
  for (int i = 0; i < N; i++) {
   sumError += abs(rx[i]-1.0f);
   sumError += abs(ry[i]-2.0f);
  }
  printf("Sum error: %f\n", sumError);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
  free(rx);
  free(ry); 
}
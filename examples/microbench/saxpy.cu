
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <algorithm>


__global__
void saxpy(int n, bool* x, float* y, int* z_first, int* z_second)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x; 
 
  //printf("blkidx: %d", blockIdx.x);
 
  //printf("tid: %d\n", i);
  //if(x[i])
  if(i<n)
  {
    //printf("here! %d", i);
    //for (int k = x[i]; k<x[i]+1; k++)
    {
      //printf("zfirst: %d\n", z_first[i]);
      z_second[i] = z_first[i] - 2.0f;
    }
    //if( i % 2 == 0)
    //  x[i] = true;
  }
  
   //y[i] = x[i]+100;
 
 
}

int main(void)
{
  int N = 512;
  int T = 32;
  // changes to for loop 
  // 4 blocks -> 4threads
  // 32 threads -> 1 for loop

  float *y,  *d_y;
  bool *x, *d_x;
  int *z_first, *d_z_first;
  int *z_second, *d_z_second;
  
  x = (bool*)malloc(N*sizeof(bool));
  y = (float*)malloc(N*sizeof(float));
  z_first = (int*)malloc(N*sizeof(int));
  z_second = (int*)malloc(N*sizeof(int));

  hipMalloc(&d_x, N*sizeof(bool)); 
  hipMalloc(&d_y, N*sizeof(float));
  hipMalloc(&d_z_first, N*sizeof(int));
  hipMalloc(&d_z_second, N*sizeof(int));

  for (int i = 0; i < N; i++) {
    if(i % 10 == 0)
      x[i] = true;
    else
      x[i] = false;
    y[i] = 2.0f;
    z_first[i] = i;
    z_second[i] = 0;
  }

  

  hipMemcpy(d_x, x, N*sizeof(bool), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_z_first, z_first, N*sizeof(int), hipMemcpyHostToDevice);
  //cudaMemset(d_z_second, 0, N*sizeof(int));
  hipMemcpy(d_z_second, z_second, N*sizeof(int), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  
  dim3 dimBlock(T);
  dim3 dimGrid(N/T);
  saxpy<<<dimGrid, dimBlock>>>(N, d_x, d_y, d_z_first, d_z_second);
  
  
  hipMemcpy(x, d_x, N*sizeof(bool), hipMemcpyDeviceToHost);
  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(z_first, d_z_first, N*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(z_second, d_z_second, N*sizeof(int), hipMemcpyDeviceToHost);

  float maxError = 0.0f;

  for (int i = 0; i < N; i++)
  {
    maxError += (z_second[i]-i+2.0f);
  }
  printf("Max error: %f\n", maxError);
  printf("Actual values are: \n");

  for (int i = 0; i < 500; i++) 
  {
        if (i%10 == 0)
    {
      printf("\n");
    }
    
    printf("%d ", z_second[i]);

  }
  printf("\n");

  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z_first);
  hipFree(d_z_second);
  free(x);
  free(y);
  free(z_first);
  free(z_second);
}


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <algorithm>

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  
  //if (i < n) 
  printf("i: %d\n", i);
  {y[i] = x[i]-y[i];}
}

int main(void)
{
  int N = 1024;//32*32*32*4;
  int T = 16;

  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(int));
  y = (float*)malloc(N*sizeof(int));

  hipMalloc(&d_x, N*sizeof(int)); 
  hipMalloc(&d_y, N*sizeof(int));

  for (int i = 0; i < N; i++) {
    x[i] = float(i);
    y[i] = 1.0f;
  }

  hipMemcpy(d_x, x, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(int), hipMemcpyHostToDevice);
  //cudaMemset((void *)d_y, 2, N * sizeof(float));7&
  

  // Perform SAXPY on 1M elements
  saxpy<<<(N/T), T>>>(N, 2.0f, d_x, d_y);

  hipMemcpy(y, d_y, N*sizeof(int), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  
  for (int i = 0; i < N; i++) {
    maxError += std::max(maxError, abs(y[i]-1.0f));
    printf("%f ", y[i]);
  }
  
  printf("Max error: %f\n", maxError);
  //printf("host_x: %ld\n", d_x);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}

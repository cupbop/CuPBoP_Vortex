#include "hip/hip_runtime.h"
// Testcase1: Evaluating thread_group and basic synchronization functions
// with tiled partitioning.

#include <hip/hip_runtime.h>
#include <stdio.h>
#include "cooperative_groups_fake.h"
// #include <hip/hip_cooperative_groups.h>


using namespace cooperative_groups;


// __global__ void cgkernel()
// {
//     printf("%d: A!\n", threadIdx.x);
//     __syncwarp();
//     printf("%d: B!\n", threadIdx.x);
// }


__global__ void cgkernel() {

    bool anyResult = __any_sync(-1, threadIdx.x);

    printf("  any result %s\n", anyResult ? "true" : "false");

    __syncwarp();
}

// __global__ void cgkernel() {
//     int laneId = threadIdx.x % warpSize;

//     // For demonstration, let’s choose a simple predicate:
//     // e.g., whether laneId is less than 16
//     bool predicate = (laneId < 16);

//     bool anyResult = __any_sync(-1, predicate);
//     bool allResult = __all_sync(-1, predicate);

//     // Print results from lane 0 of the warp to avoid too much output.
//     if (laneId == 0) {
//         printf("Within the warp:\n");
//         printf("  Any thread < 16? %s\n", anyResult ? "true" : "false");
//         printf("  All threads < 16? %s\n", allResult ? "true" : "false");
//     }

//     __syncwarp();
// }

// __global__ void cgkernel()
// {
//     thread_block block = this_thread_block();
//     thread_group g = tiled_partition(block, 16);
//     printf("%d => %d\n", block.thread_rank(), g.thread_rank());
//     g.sync();
// }

// __global__ void cgkernel()
// {
//     thread_block block = this_thread_block();
//     printf("%d\n", block.thread_rank());
//     block.sync();
//     __syncthreads();
//     thread_group g = tiled_partition(block, 16);
//     printf("%d => %d\n", block.thread_rank(), g.thread_rank());
//     g.sync();
//     printf(".\n");
// }

int main()
{
    printf("Executing Test Case 1: Testing thread_group and basic synchronization with tiled partitioning.\n");
    cgkernel<<<1, 16>>>();
    hipDeviceSynchronize();

    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    printf("Device finished successfully.\n");
    return 0;
}

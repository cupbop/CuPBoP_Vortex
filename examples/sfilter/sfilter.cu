// sfilter_cuda.cu
// CUDA port that mirrors the provided OpenCL sfilter example (with tunable block size)
// Build: nvcc -O3 -std=c++17 -o sfilter_cuda sfilter_cuda.cu
// Run:   ./sfilter_cuda -n 16 -b16x16   (or -bx 16 -by 16)

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <vector>
#include <chrono>
#include <unistd.h>
#include <cmath>

#define CUDA_CHECK(expr) do {                                     \
  hipError_t _err = (expr);                                      \
  if (_err != hipSuccess) {                                      \
    fprintf(stderr, "CUDA Error: %s at %s:%d\n",                  \
            hipGetErrorString(_err), __FILE__, __LINE__);        \
    std::abort();                                                 \
  }                                                               \
} while (0)

static bool almost_equal(float a, float b, int ulp = 4) {
  union fi_t { int i; float f; };
  fi_t fa, fb;
  fa.f = a;
  fb.f = b;
  return std::abs(fa.i - fb.i) <= ulp;
}

static void show_usage() {
  printf("Usage: sfilter_cuda [-n size] [-b WxH | -bx W -by H] [-h]\n");
  printf("  -n   : matrix size (N x N), N >= 3 (default 16)\n");
  printf("  -b   : block size in 'WxH' format (e.g., -b32x8)\n");
  printf("  -bx  : blockDim.x (e.g., -bx 32)\n");
  printf("  -by  : blockDim.y (e.g., -by 8)\n");
  printf("  -h   : help\n");
}

static int size_ = 16;
static int block_x = 16;
static int block_y = 16;

static void parse_b_arg(const char* s) {
  // parse "WxH"
  int w = 0, h = 0;
  if (std::sscanf(s, "%dx%d", &w, &h) == 2 && w > 0 && h > 0) {
    block_x = w; block_y = h;
  } else {
    fprintf(stderr, "Invalid -b format. Use -b32x8\n");
    std::exit(-1);
  }
}

static void parse_args(int argc, char **argv) {
  int c;
  // getopt string includes options with arguments: n:, b: and long-ish bx/by handled manually
  while ((c = getopt(argc, argv, "n:b:h")) != -1) {
    switch (c) {
      case 'n':
        size_ = std::atoi(optarg);
        break;
      case 'b':
        parse_b_arg(optarg);
        break;
      case 'h':
        show_usage();
        std::exit(0);
      default:
        show_usage();
        std::exit(-1);
    }
  }
  // handle -bx and -by if present (manual scan)
  for (int i = 1; i < argc; ++i) {
    if (std::strcmp(argv[i], "-bx") == 0 && i + 1 < argc) {
      block_x = std::atoi(argv[i + 1]);
    }
    if (std::strcmp(argv[i], "-by") == 0 && i + 1 < argc) {
      block_y = std::atoi(argv[i + 1]);
    }
  }
  if (size_ < 3) {
    fprintf(stderr, "Error: input size must be >= 3\n");
    std::exit(-1);
  }
  if (block_x <= 0 || block_y <= 0) {
    fprintf(stderr, "Error: invalid block size (%d x %d)\n", block_x, block_y);
    std::exit(-1);
  }
}

// m0 m1 m2
// m3 m4 m5
// m6 m7 m8
// __global__ void sfilter_kernel(const float* src, //(removed __restrict__)
//                                float* dst, //(removed __restrict__)
//                                long long ldc,
//                                float m0,float m1,float m2,
//                                float m3,float m4,float m5,
//                                float m6,float m7,float m8)
// {
//   // Emulate OpenCL's global_offset = {1,1} with +1 here
//   const long long x0 = blockIdx.x * blockDim.x + threadIdx.x;
//   const long long y0 = blockIdx.y * blockDim.y + threadIdx.y;
//   const long long x = x0 + 1;
//   const long long y = y0 + 1;

//   // Only compute interior pixels [1..ldc-2] just like global size {size-2,size-2}
//   if (x >= ldc - 1 || y >= ldc - 1) return;

//   const long long addr = x + y * ldc;

//   float i0 = src[addr-1-1*ldc]*m0;
//   float i1 = src[addr+0-1*ldc]*m1;
//   float i2 = src[addr+1-1*ldc]*m2;
//   float i3 = src[addr-1+0*ldc]*m3;
//   float i4 = src[addr+0+0*ldc]*m4;
//   float i5 = src[addr+1+0*ldc]*m5;
//   float i6 = src[addr-1+1*ldc]*m6;
//   float i7 = src[addr+0+1*ldc]*m7;
//   float i8 = src[addr+1+1*ldc]*m8;

//   dst[addr] = i0 + i1 + i2 + i3 + i4 + i5 + i6 + i7 + i8;
// }

__global__ void sfilter_kernel(const float* __restrict__ src,
                               float* __restrict__ dst,
                               int ldc,           // int로 충분
                               float m0,float m1,float m2,
                               float m3,float m4,float m5,
                               float m6,float m7,float m8)
{
  // 작업 영역 크기 (내부만 계산)
  const int work_x = ldc - 2;
  const int work_y = ldc - 2;

  // 오프셋 적용 전의 “작업 좌표”
  // blockidx.y 0-61 
  // threadidx.x 0-63

  const int x0 = blockIdx.x * blockDim.x + threadIdx.x; // x0(0-64)
  const int y0 = blockIdx.y * blockDim.y + threadIdx.y; // y0(62)

    // 실제 좌표 = 내부 시작점 (1,1) 에서 오프셋 +1
  const int x = x0 + 1;
  const int y = y0 + 1;

  const int addr = x + y * ldc;
  if (x0 == 0)
  {
    printf("blockidx.y= %d", blockIdx.y);
  }
  //printf("blockdim.x: %d, blockDim.y: %d", blockDim.x, blockDim.y);
  //printf("griddim.x: %d griddim.y:  %d", gridDim.x, gridDim.y);
  // if (y0 == 16)
  // {
  //      printf("blockIdx.x: %d, threadIdx.x: %d", blockIdx.x, threadIdx.x);
  //       printf("blockIdx.y: %d, threadIdx.y: %d", blockIdx.y, threadIdx.y);
  //       printf("x0=%d, y0=%d", x0, y0); 
  // }
  // if (addr > 1089 && addr <1150)
  // {
  //   printf("blockIdx.x: %d, threadIdx.x: %d", blockIdx.x, threadIdx.x);
  //   printf("blockIdx.y: %d, threadIdx.y: %d", blockIdx.y, threadIdx.y);
  //   printf("x0=%d, y0=%d", x0, y0);
  // }

  // work 범위 밖이면 리턴 (오프셋 적용 前 기준)
  if (x0 >= work_x || y0 >= work_y) return;




  float i0 = src[addr-1-1*ldc]*m0;
  float i1 = src[addr+0-1*ldc]*m1;
  float i2 = src[addr+1-1*ldc]*m2;
  float i3 = src[addr-1+0*ldc]*m3;
  float i4 = src[addr+0+0*ldc]*m4;
  float i5 = src[addr+1+0*ldc]*m5;
  float i6 = src[addr-1+1*ldc]*m6;
  float i7 = src[addr+0+1*ldc]*m7;
  float i8 = src[addr+1+1*ldc]*m8;

  dst[addr] = i0 + i1 + i2 + i3 + i4 + i5 + i6 + i7 + i8;
}

int main(int argc, char** argv) {
  parse_args(argc, argv);

  printf("input size=%d\n", size_);
  printf("block size=%dx%d\n", block_x, block_y);

  const long long ldc = size_;
  const size_t n = (size_t)size_ * size_;
  const size_t nbytes = n * sizeof(float);

  // Host buffers
  std::vector<float> src_h(n);
  std::vector<float> dst_h(n, 0.0f);
  std::vector<float> ref_vec(n, 0.0f);

  // Init host input
  for (size_t i = 0; i < n; ++i) {
    src_h[i] = static_cast<float>(rand()) / RAND_MAX * 100.0f;
  }

  // Masks (same defaults as OpenCL host)
  float m0 = 1.0f, m1 = 1.0f, m2 = 1.0f;
  float m3 = 1.0f, m4 = 1.0f, m5 = 1.0f;
  float m6 = 1.0f, m7 = 1.0f, m8 = 1.0f;

  // Reference computation (OpenCL style launch mapping)
  {
    const size_t global_offset[2]    = {1, 1};
    const size_t global_work_size[2] = {(size_t)(size_ - 2), (size_t)(size_ - 2)};
    const size_t local_work_size[2]  = {1, 1};

    size_t num_groups_y = global_work_size[1] / local_work_size[1];
    size_t num_groups_x = global_work_size[0] / local_work_size[0];
    for (size_t workgroup_id_y = 0; workgroup_id_y < num_groups_y; ++workgroup_id_y) {
      for (size_t workgroup_id_x = 0; workgroup_id_x < num_groups_x; ++workgroup_id_x) {
        for (size_t local_id_y = 0; local_id_y < local_work_size[1]; ++local_id_y) {
          for (size_t local_id_x = 0; local_id_x < local_work_size[0]; ++local_id_x) {
            int x = global_offset[0] + local_work_size[0] * workgroup_id_x + local_id_x;
            int y = global_offset[1] + local_work_size[1] * workgroup_id_y + local_id_y;
            long long addr = x + (long long)y * ldc;
            float i0 = src_h.at(addr-1-1*ldc)*m0;
            float i1 = src_h.at(addr+0-1*ldc)*m1;
            float i2 = src_h.at(addr+1-1*ldc)*m2;
            float i3 = src_h.at(addr-1+0*ldc)*m3;
            float i4 = src_h.at(addr+0+0*ldc)*m4;
            float i5 = src_h.at(addr+1+0*ldc)*m5;
            float i6 = src_h.at(addr-1+1*ldc)*m6;
            float i7 = src_h.at(addr+0+1*ldc)*m7;
            float i8 = src_h.at(addr+1+1*ldc)*m8;
            float v = i0 + i1 + i2 + i3 + i4 + i5 + i6 + i7 + i8;
            ref_vec.at(addr) = v;
          }
        }
      }
    }
  }

  // Device buffers
  float *src_d = nullptr, *dst_d = nullptr;
  CUDA_CHECK(hipMalloc(&src_d, nbytes));
  CUDA_CHECK(hipMalloc(&dst_d, nbytes));

  // Upload
  CUDA_CHECK(hipMemcpy(src_d, src_h.data(), nbytes, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemset(dst_d, 0, nbytes));

  // Launch config:
  // domain = interior region size_ - 2 by size_ - 2 (because of +1 offset in kernel)
  const int work_x = size_ - 2;
  const int work_y = size_ - 2;

  dim3 block(block_x, block_y, 1);
  dim3 grid((work_x + block_x - 1) / block_x,
            (work_y + block_y - 1) / block_y,
            1);

  printf("enqueue kernel\n");
  printf("global_work_size=[%d, %d]  block=[%d, %d]  grid=[%d, %d]\n",
         work_x, work_y, block_x, block_y, grid.x, grid.y);

  auto t0 = std::chrono::high_resolution_clock::now();
  sfilter_kernel<<<grid, block>>>(src_d, dst_d, size_,
                                  m0,m1,m2,m3,m4,m5,m6,m7,m8);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
  auto t1 = std::chrono::high_resolution_clock::now();
  double elapsed_ms = std::chrono::duration_cast<std::chrono::milliseconds>(t1 - t0).count();
  printf("Elapsed time: %g ms\n", elapsed_ms);

  // Download
  CUDA_CHECK(hipMemcpy(dst_h.data(), dst_d, nbytes, hipMemcpyDeviceToHost));

  // Verify
  int errors = 0;
  for (size_t i = 0; i < n; ++i) {
    if (!almost_equal(dst_h[i], ref_vec[i])) {
      if (errors < 100) {
        printf("*** error: [%zu] expected=%f, actual=%f\n", i, ref_vec[i], dst_h[i]);
      }
      ++errors;
    }
  }
  if (errors == 0) {
    printf("PASSED!\n");
  } else {
    printf("FAILED! - %d errors\n", errors);
  }

  // Cleanup
  hipFree(src_d);
  hipFree(dst_d);
  return errors;
}
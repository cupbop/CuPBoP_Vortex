#include "hip/hip_runtime.h"
//========================================================================================================================================================================================================200
//	findRangeK function
//========================================================================================================================================================================================================200

__global__ void
findRangeK(	long height,

			knode *knodesD,
			long knodes_elem,

			long *currKnodeD,
			long *offsetD,
			long *lastKnodeD,
			long *offset_2D,
			int *startD,
			int *endD,
			int *RecstartD,
			int *ReclenD)
{

	// private thread IDs
	int thid = threadIdx.x;
	int bid = blockIdx.x;

	// ???

	//printf("currKnodeD[%d] = %ld, lastKnodeD[%d] = %ld\n", bid, currKnodeD[bid], bid, lastKnodeD[bid]);
	// if (thid == 0) {
	// 	printf("startD[%d] = %d, endD[%d] = %d\n", bid, startD[bid], bid, endD[bid]);
	// }

	int i;
	for(i = 0; i < height; i++){

		if((knodesD[currKnodeD[bid]].keys[thid] <= startD[bid]) && (knodesD[currKnodeD[bid]].keys[thid+1] > startD[bid])){
			//printf("knodesD[%ld].keys[%d] = %d, knodesD[%ld].keys[%d+1] = %d, startD[%d] = %d\n", currKnodeD[bid], thid, knodesD[currKnodeD[bid]].keys[thid], currKnodeD[bid], thid, knodesD[currKnodeD[bid]].keys[thid+1], bid, startD[bid]);
			// this conditional statement is inserted to avoid crush due to but in original code
			// "offset[bid]" calculated below that later addresses part of knodes goes outside of its bounds cause segmentation fault
			// more specifically, values saved into knodes->indices in the main function are out of bounds of knodes that they address
			if(knodesD[currKnodeD[bid]].indices[thid] < knodes_elem){
				offsetD[bid] = knodesD[currKnodeD[bid]].indices[thid];
				//printf("offsetD[%d] = %ld\n knodesD[%ld].indices[%d] = %d, bid = %d, knodes_elem = %ld\n", bid, offsetD[bid], currKnodeD[bid], thid, knodesD[currKnodeD[bid]].indices[thid], bid, knodes_elem);
			}
		}
		if((knodesD[lastKnodeD[bid]].keys[thid] <= endD[bid]) && (knodesD[lastKnodeD[bid]].keys[thid+1] > endD[bid])){
			// this conditional statement is inserted to avoid crush due to but in original code
			// "offset_2[bid]" calculated below that later addresses part of knodes goes outside of its bounds cause segmentation fault
			// more specifically, values saved into knodes->indices in the main function are out of bounds of knodes that they address
			if(knodesD[lastKnodeD[bid]].indices[thid] < knodes_elem){
				offset_2D[bid] = knodesD[lastKnodeD[bid]].indices[thid];
			}
		}
		__syncthreads();

		// set for next tree level
		if(thid==0){
			currKnodeD[bid] = offsetD[bid];
			lastKnodeD[bid] = offset_2D[bid];
		}
		__syncthreads();
	}

	// Find the index of the starting record
	if(knodesD[currKnodeD[bid]].keys[thid] == startD[bid]){
		RecstartD[bid] = knodesD[currKnodeD[bid]].indices[thid];
	}
	__syncthreads();

	// Find the index of the ending record
	if(knodesD[lastKnodeD[bid]].keys[thid] == endD[bid]){
		ReclenD[bid] = knodesD[lastKnodeD[bid]].indices[thid] - RecstartD[bid]+1;
	}

}

//========================================================================================================================================================================================================200
//	End
//========================================================================================================================================================================================================200

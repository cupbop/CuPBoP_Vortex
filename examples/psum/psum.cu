// parallel_sum.cu
// nvcc -O2 -std=c++17 parallel_sum.cu -o parallel_sum
#include <cstdio>
#include <cstdlib>
#include <cstdint>
#include <cassert>
#include <cmath>
#include <cstring>
#include <chrono>
#include <vector>
#include <algorithm>
#include <numeric>
#include <unistd.h>   // getopt

#include <hip/hip_runtime.h>

#define FLOAT_ULP 16

#define CUDA_CHECK(expr)                                                         \
  do {                                                                           \
    hipError_t _err = (expr);                                                   \
    if (_err != hipSuccess) {                                                   \
      fprintf(stderr, "CUDA Error: %s -> %d (%s)\n", #expr, (int)_err,           \
              hipGetErrorString(_err));                                         \
      std::exit(-1);                                                             \
    }                                                                            \
  } while (0)

static inline bool compare_equal(float a, float b) {
  union fi_t { float f; int32_t i; };
  fi_t fa{.f=a}, fb{.f=b};
  auto d = std::abs(fa.i - fb.i);
  return (d/fa.i) <= 1e-3;
}

static float computeParallelSumCPU(const float* A, int N) {
  float sum = 0.f;
  for (int i = 0; i < N; ++i) sum += A[i];
  return sum;
}

// ---------------- CUDA kernel: 1:1 대응 ----------------
__global__ void parallelSumKernel(const float* __restrict__ input,
                                  float* __restrict__ output,
                                  int N) {
  extern __shared__ float temp[];  // dynamic shared memory (OpenCL __local)
  int local_id   = threadIdx.x;               // get_local_id(0)
  int local_size = blockDim.x;                // get_local_size(0)
  int global_id  = blockIdx.x * blockDim.x + threadIdx.x; // get_global_id(0)

  // Load into shared (local) memory
  if (global_id < N) {
    temp[local_id] = input[global_id];
  } else {
    temp[local_id] = 0.0f;
  }
  __syncthreads(); // barrier(CLK_LOCAL_MEM_FENCE)

  // Reduction in shared memory
  for (unsigned int stride = local_size >> 1; stride > 0; stride >>= 1) {
    if (local_id < (int)stride) {
      temp[local_id] += temp[local_id + stride];
    }
    __syncthreads();
  }

  // Write per-block result
  if (local_id == 0) {
    // get_group_id(0) == blockIdx.x
    output[blockIdx.x] = temp[0];
  }
}

// ---------------- Host main ----------------
int main(int argc, char** argv) {
  // Default params match the OpenCL host
  size_t size = 16;       // total inputs (N)
  size_t local_size = 8;  // block size

  // Parse args: -n size, -l local_size, -h
  int c;
  while ((c = getopt(argc, argv, "n:l:h")) != -1) {
    switch (c) {
      case 'n': size = std::max(1, atoi(optarg)); break;
      case 'l': local_size = std::max(1, atoi(optarg)); break;
      case 'h':
      default:
        printf("Usage: [-n size] [-l local_size] [-h]\n");
        return 0;
    }
  }

  printf("input size=%zu, local size=%zu\n", size, local_size);
  if ((size / local_size) * local_size != size) {
    printf("Error: input size must be a multiple of %zu\n", local_size);
    return -1;
  }

  const uint32_t num_inputs  = static_cast<uint32_t>(size);
  const uint32_t num_outputs = static_cast<uint32_t>(size / local_size);

  // Show device
  hipDeviceProp_t prop{};
  int device = 0;
  CUDA_CHECK(hipGetDevice(&device));
  CUDA_CHECK(hipGetDeviceProperties(&prop, device));
  printf("Using device: %s\n", prop.name);

  // Host buffers
  std::vector<float> h_a(num_inputs);
  std::vector<float> h_c(num_outputs);

  // Random inputs (OpenCL 코드와 동치의 임의값 생성)
  for (uint32_t i = 0; i < num_inputs; ++i) {
    h_a[i] = static_cast<float>(rand()) / RAND_MAX;
  }

  // Device buffers
  float *d_a = nullptr, *d_c = nullptr;
  size_t i_nbytes = num_inputs * sizeof(float);
  size_t o_nbytes = num_outputs * sizeof(float);
  CUDA_CHECK(hipMalloc((void**)&d_a, i_nbytes));
  CUDA_CHECK(hipMalloc((void**)&d_c, o_nbytes));

  // Upload
  CUDA_CHECK(hipMemcpy(d_a, h_a.data(), i_nbytes, hipMemcpyHostToDevice));

  // Grid/block & dynamic shared mem size
  dim3 block(static_cast<unsigned int>(local_size), 1, 1);
  dim3 grid (static_cast<unsigned int>(num_outputs), 1, 1); // size/local_size
  size_t smemSize = local_size * sizeof(float);             // __local float* temp

  // Launch + timing (NDRange: global=size, local=local_size 동일하게 매핑)
  auto t0 = std::chrono::high_resolution_clock::now();
  parallelSumKernel<<<grid, block, smemSize>>>(d_a, d_c, (int)num_inputs);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
  auto t1 = std::chrono::high_resolution_clock::now();
  double elapsed_ms =
      std::chrono::duration_cast<std::chrono::milliseconds>(t1 - t0).count();
  printf("Elapsed time: %lg ms\n", elapsed_ms);

  // Download
  CUDA_CHECK(hipMemcpy(h_c.data(), d_c, o_nbytes, hipMemcpyDeviceToHost));

  // Verify (OpenCL 호스트와 동일한 검증 방식)
  int errors = 0;
  float result = computeParallelSumCPU(h_c.data(), num_outputs);
  float gold   = computeParallelSumCPU(h_a.data(), num_inputs);

  if (!compare_equal(result, gold)) {
    printf("*** error: expected=%f, actual=%f", gold, result);
    for (uint32_t i = 0; i < num_outputs; ++i) {
      printf(", output[%u]=%f", i, h_c[i]);
    }
    printf("\n");
    errors = 1;
  }

  if (errors != 0) printf("FAILED! - %d errors\n", errors);
  else             printf("PASSED!\n");

  // Cleanup
  CUDA_CHECK(hipFree(d_a));
  CUDA_CHECK(hipFree(d_c));
  return errors;
}
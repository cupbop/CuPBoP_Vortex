// sgemm2.cu
// nvcc -O2 -std=c++17 sgemm2.cu -o sgemm2
#include <cstdio>
#include <cstdlib>
#include <cstdint>
#include <cassert>
#include <cstring>
#include <cmath>
#include <vector>
#include <chrono>
#include <unistd.h>   // getopt

#include <hip/hip_runtime.h>

// ---- [common.h] 통합 ----
#define TILE_SIZE 16
#ifndef TYPE
#define TYPE float
#endif
// -------------------------

#define FLOAT_ULP 6

#define CUDA_CHECK(expr)                                                         \
  do {                                                                           \
    hipError_t _err = (expr);                                                   \
    if (_err != hipSuccess) {                                                   \
      fprintf(stderr, "CUDA Error: %s -> %d (%s)\n", #expr, (int)_err,           \
              hipGetErrorString(_err));                                         \
      std::exit(-1);                                                             \
    }                                                                            \
  } while (0)

// OpenCL 호스트 코드와 동일 로직의 비교/생성 유틸
template <typename T> struct Comparator {};

template <> struct Comparator<int> {
  static const char* type_str() { return "integer"; }
  static int generate() { return std::rand(); }
  static bool compare(int a, int b, int index, int errors) {
    if (a != b) {
      if (errors < 100)
        std::printf("*** error: [%d] expected=%d, actual=%d\n", index, a, b);
      return false;
    }
    return true;
  }
};

template <> struct Comparator<float> {
  static const char* type_str() { return "float"; }
  static float generate() { return static_cast<float>(std::rand()) / RAND_MAX; }
  static bool compare(float a, float b, int index, int errors) {
    union fi_t { float f; int32_t i; };
    fi_t fa, fb;
    fa.f = a; fb.f = b;
    auto d = std::abs(fa.i - fb.i);
    if (d > FLOAT_ULP) {
      if (errors < 100)
        std::printf("*** error: [%d] expected=%f, actual=%f\n", index, a, b);
      return false;
    }
    return true;
  }
};

// CPU 참조(SGEMM) — 제공된 OpenCL 호스트 코드의 인덱싱을 그대로 사용
static void sgemm_cpu(TYPE *C, const TYPE* A, const TYPE *B, int M, int N, int K) {
  for (int m = 0; m < M; ++m) {
    for (int n = 0; n < N; ++n) {
      TYPE acc = 0;
      for (int k = 0; k < K; ++k) {
        acc += A[k * M + m] * B[n * K + k];
      }
      C[n * M + m] = acc;
    }
  }
}

// ---------------- CUDA Kernel: OpenCL sgemm2와 1:1 논리 ----------------
__global__ void sgemm2_kernel(const TYPE* __restrict__ A,
                              const TYPE* __restrict__ B,
                              TYPE* __restrict__ C,
                              unsigned int N)
{
  // OpenCL 매핑:
  // globalCol = get_global_id(0)  == blockIdx.x * blockDim.x + threadIdx.x
  // globalRow = get_global_id(1)  == blockIdx.y * blockDim.y + threadIdx.y
  // localCol  = get_local_id(0)   == threadIdx.x
  // localRow  = get_local_id(1)   == threadIdx.y
  int globalCol = blockIdx.x * blockDim.x + threadIdx.x;
  int globalRow = blockIdx.y * blockDim.y + threadIdx.y;
  int localCol  = threadIdx.x;
  int localRow  = threadIdx.y;

  // __local TYPE localA[TILE_SIZE][TILE_SIZE];
  // __local TYPE localB[TILE_SIZE][TILE_SIZE];
  __shared__ TYPE localA[TILE_SIZE][TILE_SIZE];
  __shared__ TYPE localB[TILE_SIZE][TILE_SIZE];

  TYPE sum = 0;

  // for (int k = 0; k < N; k += TILE_SIZE)
  for (int k = 0; k < (int)N; k += TILE_SIZE) {
    // localA[localRow][localCol] = A[globalRow * N + (k + localCol)];
    // localB[localRow][localCol] = B[(k + localRow) * N + globalCol];
    // localA[localRow][localCol] = A[globalRow * N + (k + localCol)];
    // localB[localRow][localCol] = B[(k + localRow) * N + globalCol];
    // A 타일: 행 m = globalRow, 열 k = k0 + localCol  → A[(k0+localCol)*M + globalRow]
    localA[localRow][localCol] = A[(k + localCol) * N + globalRow];

    // B 타일: 행 k = k0 + localRow, 열 n = globalCol → B[globalCol*K + (k0+localRow)]
    localB[localRow][localCol] = B[globalCol * N + (k + localRow)];

    __syncthreads(); // barrier(CLK_LOCAL_MEM_FENCE)

    // for (int j = 0; j < TILE_SIZE; j++) sum += localA[localRow][j] * localB[j][localCol];
    #pragma unroll
    for (int j = 0; j < TILE_SIZE; ++j) {
      sum += localA[localRow][j] * localB[j][localCol];
    }

    __syncthreads(); // barrier(CLK_LOCAL_MEM_FENCE) — Deadlock case 방지 코드 위치 동일
  }

  // C[globalRow * N + globalCol] = sum;
  //임시
  // C[globalRow * N + globalCol] = sum;
  C[globalCol * N + globalRow] = sum;
}

// ---------------- Host main (OpenCL 호스트 로직 1:1 매핑) ----------------
int main(int argc, char** argv) {
  std::srand(0xC0FFEE);

  uint32_t size = 16; // N
  // 옵션 파싱: -n size
  int opt;
  while ((opt = ::getopt(argc, argv, "n:h")) != -1) {
    switch (opt) {
      case 'n': size = std::max(1, atoi(optarg)); break;
      case 'h':
      default:
        std::printf("Usage: [-n size] [-h]\n");
        return 0;
    }
  }

  // TILE_SIZE 배수 체크 (OpenCL 호스트와 동일)
  std::printf("Matrix size=%ux%u, tile size=%dx%d\n",
              size, size, TILE_SIZE, TILE_SIZE);
  if ((size / TILE_SIZE) * TILE_SIZE != size) {
    std::printf("Error: matrix size must be a multiple of %d\n", TILE_SIZE);
    return -1;
  }

  const uint32_t N = size;
  const uint32_t size_sq = N * N;
  const size_t nbytes = size_sq * sizeof(TYPE);

  // 디바이스 정보 출력
  int dev = 0;
  CUDA_CHECK(hipGetDevice(&dev));
  hipDeviceProp_t prop{};
  CUDA_CHECK(hipGetDeviceProperties(&prop, dev));
  std::printf("Using device: %s\n", prop.name);

  // 호스트 버퍼
  std::vector<TYPE> h_a(size_sq), h_b(size_sq), h_c(size_sq), h_ref(size_sq);

  // 입력 생성 (OpenCL Comparator.generate() 동등)
  for (uint32_t i = 0; i < size_sq; ++i) {
    h_a[i] = Comparator<TYPE>::generate();
    h_b[i] = Comparator<TYPE>::generate();
  }

  // 디바이스 메모리
  TYPE *d_a = nullptr, *d_b = nullptr, *d_c = nullptr;
  CUDA_CHECK(hipMalloc((void**)&d_a, nbytes));
  CUDA_CHECK(hipMalloc((void**)&d_b, nbytes));
  CUDA_CHECK(hipMalloc((void**)&d_c, nbytes));

  // 업로드
  std::printf("Upload source buffers\n");
  CUDA_CHECK(hipMemcpy(d_a, h_a.data(), nbytes, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_b, h_b.data(), nbytes, hipMemcpyHostToDevice));

  // 실행 파라미터 (OpenCL: global={N,N}, local={TILE_SIZE,TILE_SIZE})
  dim3 block(TILE_SIZE, TILE_SIZE);
  dim3 grid (N / TILE_SIZE, N / TILE_SIZE);
  std::printf("global_work_size=%ux%u, local_work_size=%dx%d\n",
              N, N, TILE_SIZE, TILE_SIZE);

  // 커널 실행 & 타이밍
  auto t0 = std::chrono::high_resolution_clock::now();
  sgemm2_kernel<<<grid, block>>>(d_a, d_b, d_c, N);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
  auto t1 = std::chrono::high_resolution_clock::now();
  double elapsed_ms =
      std::chrono::duration_cast<std::chrono::milliseconds>(t1 - t0).count();
  std::printf("Elapsed time: %lg ms\n", elapsed_ms);

  // 다운로드
  std::printf("Download destination buffer\n");
  CUDA_CHECK(hipMemcpy(h_c.data(), d_c, nbytes, hipMemcpyDeviceToHost));

  // 검증 (OpenCL 호스트와 동일한 CPU sgemm)
  std::printf("Verify result\n");
  sgemm_cpu(h_ref.data(), h_a.data(), h_b.data(), N, N, N);
  int errors = 0;
  for (uint32_t i = 0; i < size_sq; ++i) {
    if (!Comparator<TYPE>::compare(h_c[i], h_ref[i], i, errors)) {
      ++errors;
    }
  }
  if (errors != 0) std::printf("FAILED! - %d errors\n", errors);
  else             std::printf("PASSED!\n");

  // 정리
  CUDA_CHECK(hipFree(d_a));
  CUDA_CHECK(hipFree(d_b));
  CUDA_CHECK(hipFree(d_c));
  return errors;
}
// psort.cu
// nvcc -O2 psort.cu -o psort
#include <cstdio>
#include <cstdlib>
#include <cassert>
#include <cmath>
#include <chrono>
#include <cstring>
#include <unistd.h>

#include <hip/hip_runtime.h>

#define KERNEL0_NAME "psorti"
#define KERNEL1_NAME "psortf"

#define CUDA_CHECK(expr)                                                     \
  do {                                                                       \
    hipError_t _err = (expr);                                               \
    if (_err != hipSuccess) {                                               \
      fprintf(stderr, "CUDA Error: %s returned %d (%s)\n", #expr,            \
              (int)_err, hipGetErrorString(_err));                          \
      cleanup();                                                             \
      exit(-1);                                                              \
    }                                                                        \
  } while (0)

static int size_ = 64;
static bool float_enable = false;

// Host buffers
static int*   h_ai = nullptr;   // used for int mode
static int*   h_ci = nullptr;
static float* h_af = nullptr;   // used for float mode
static float* h_cf = nullptr;

// Device buffers
static int*   d_ai = nullptr;
static int*   d_ci = nullptr;
static float* d_af = nullptr;
static float* d_cf = nullptr;

static void cleanup() {
  if (d_ai) CUDA_CHECK(hipFree(d_ai));
  if (d_ci) CUDA_CHECK(hipFree(d_ci));
  if (d_af) CUDA_CHECK(hipFree(d_af));
  if (d_cf) CUDA_CHECK(hipFree(d_cf));

  if (h_ai) free(h_ai);
  if (h_ci) free(h_ci);
  if (h_af) free(h_af);
  if (h_cf) free(h_cf);
}

static void show_usage() {
  printf("Usage: [-f] [-n size] [-h]\n");
}

static void parse_args(int argc, char** argv) {
  int c;
  while ((c = getopt(argc, argv, "fn:h")) != -1) {
    switch (c) {
      case 'f': float_enable = true; break;
      case 'n': size_ = atoi(optarg); break;
      case 'h': show_usage(); exit(0);
      default: show_usage(); exit(-1);
    }
  }
  printf("Workload size=%d, mode=%s\n", size_, float_enable ? "float" : "int");
}

// -------------------- CUDA kernels --------------------
__global__ void psorti_kernel(const int* __restrict__ in,
                              int* __restrict__ out,
                              int n) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= n) return;

  int ref = in[gid];
  int pos = 0;
  // same logic as OpenCL: pos += (cur < ref) || ((cur == ref) && (i < gid));
  for (int i = 0; i < n; ++i) {
    int cur = in[i];
    pos += (cur < ref) || ((cur == ref) && (i < gid));
  }
  out[pos] = ref;
}

__global__ void psortf_kernel(const float* __restrict__ in,
                              float* __restrict__ out,
                              int n) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= n) return;

  float ref = in[gid];
  int pos = 0;
  for (int i = 0; i < n; ++i) {
    float cur = in[i];
    pos += (cur < ref) || ((cur == ref) && (i < gid));
  }
  out[pos] = ref;
}

// -------------------- main --------------------
int main(int argc, char** argv) {
  parse_args(argc, argv);

  // Allocate & initialize host data (match OpenCL behavior)
  if (float_enable) {
    size_t nbytes = size_ * sizeof(float);
    h_af = (float*)malloc(nbytes);
    h_cf = (float*)malloc(nbytes);
    if (!h_af || !h_cf) { fprintf(stderr, "Host alloc failed\n"); cleanup(); return -1; }

    for (int i = 0; i < size_; ++i) {
      float value = sinf(i) * sinf(i);
      h_af[i] = value;
      printf("*** [%d]: %f\n", i, value);
    }
  } else {
    size_t nbytes = size_ * sizeof(int);
    h_ai = (int*)malloc(nbytes);
    h_ci = (int*)malloc(nbytes);
    if (!h_ai || !h_ci) { fprintf(stderr, "Host alloc failed\n"); cleanup(); return -1; }

    for (int i = 0; i < size_; ++i) {
      int value = (int)(size_ * sinf(i)); // identical to given OpenCL code
      h_ai[i] = value;
      printf("*** [%d]: %d\n", i, value);
    }
  }

  // Device alloc & upload
  if (float_enable) {
    size_t nbytes = size_ * sizeof(float);
    CUDA_CHECK(hipMalloc((void**)&d_af, nbytes));
    CUDA_CHECK(hipMalloc((void**)&d_cf, nbytes));
    CUDA_CHECK(hipMemcpy(d_af, h_af, nbytes, hipMemcpyHostToDevice));
  } else {
    size_t nbytes = size_ * sizeof(int);
    CUDA_CHECK(hipMalloc((void**)&d_ai, nbytes));
    CUDA_CHECK(hipMalloc((void**)&d_ci, nbytes));
    CUDA_CHECK(hipMemcpy(d_ai, h_ai, nbytes, hipMemcpyHostToDevice));
  }

  // Launch
  int block = 256;
  int grid  = (size_ + block - 1) / block;

  printf("Execute the kernel\n");
  printf("Workload size=%d\n", size_);

  auto time_start = std::chrono::high_resolution_clock::now();
  if (float_enable) {
    psortf_kernel<<<grid, block>>>(d_af, d_cf, size_);
  } else {
    psorti_kernel<<<grid, block>>>(d_ai, d_ci, size_);
  }
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
  auto time_end = std::chrono::high_resolution_clock::now();
  double elapsed =
      std::chrono::duration_cast<std::chrono::milliseconds>(time_end - time_start).count();
  printf("Elapsed time: %lg ms\n", elapsed);

  // Download
  if (float_enable) {
    size_t nbytes = size_ * sizeof(float);
    CUDA_CHECK(hipMemcpy(h_cf, d_cf, nbytes, hipMemcpyDeviceToHost));
  } else {
    size_t nbytes = size_ * sizeof(int);
    CUDA_CHECK(hipMemcpy(h_ci, d_ci, nbytes, hipMemcpyDeviceToHost));
  }

  // Print results (like OpenCL host code)
  printf("Verify result\n");
  if (float_enable) {
    for (int i = 0; i < size_; ++i) {
      printf("*** [%d]: %f\n", i, h_cf[i]);
    }
  } else {
    for (int i = 0; i < size_; ++i) {
      printf("*** [%d]: %d\n", i, h_ci[i]);
    }
  }

  // Verify correctness (1:1 with original logic)
  int errors = 0;
  if (float_enable) {
    for (int i = 0; i < size_; ++i) {
      int pos = 0;
      float ref = h_af[i];
      for (int j = 0; j < size_; ++j) {
        float cur = h_af[j];
        pos += (cur < ref) || (cur == ref && j < i);
      }
      float value = h_cf[pos];
      if (value != ref) {
        if (errors < 100) {
          printf("*** error: [%d] expected=%f, actual=%f\n", pos, ref, value);
        }
        ++errors;
      }
    }
  } else {
    for (int i = 0; i < size_; ++i) {
      int pos = 0;
      int ref = h_ai[i];
      for (int j = 0; j < size_; ++j) {
        int cur = h_ai[j];
        pos += (cur < ref) || (cur == ref && j < i);
      }
      int value = h_ci[pos];
      if (value != ref) {
        if (errors < 100) {
          printf("*** error: [%d] expected=%d, actual=%d\n", pos, ref, value);
        }
        ++errors;
      }
    }
  }

  if (errors == 0) {
    printf("PASSED!\n");
  } else {
    printf("FAILED! - %d errors\n", errors);
  }

  cleanup();
  return errors;
}
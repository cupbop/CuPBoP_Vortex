// stencil_parboil_cuda.cu — CUDA port with Parboil-style file I/O & gold check
// Build: nvcc -O2 -std=c++17 -arch=sm_70 -o stencil_cuda stencil_parboil_cuda.cu
// Run  : ./stencil_cuda nx ny nz t -i input.bin [-o out.bin]
//
// Semantics match the OpenCL kernel:
//   Anext[idx] = c0 * A0[idx] + c1 * (6-neighborhood of A0[idx])
// Update domain: interior (1..nx-2, 1..ny-2, 1..nz-2)
// Launch mirrors OpenCL rounding on X (tx=128), with +1 offset emulated.

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cstdint>
#include <cstring>
#include <chrono>
#include <vector>
#include <string>
#include <cmath>
#include <unistd.h>   // getopt

#define CUDA_CHECK(call)                                                 \
  do {                                                                   \
    hipError_t _err = (call);                                           \
    if (_err != hipSuccess) {                                           \
      fprintf(stderr, "CUDA Error: %s (%d) at %s:%d\n",                  \
              hipGetErrorString(_err), (int)_err, __FILE__, __LINE__);  \
      exit(1);                                                           \
    }                                                                    \
  } while (0)

// ---------- I/O helpers (Parboil-style) ----------
static int read_data(float *A0, int nx, int ny, int nz, FILE *fp) {
  const size_t total = (size_t)nx * ny * nz;
  size_t got = fread(A0, sizeof(float), total, fp);
  return (got == total) ? 0 : -1;
}
static std::string replace_ext(const std::string& path, const char* ext) {
  size_t p = path.find_last_of('.');
  if (p == std::string::npos) return path + ext;
  return path.substr(0, p) + ext;
}
static float* read_gold_file(const char* filename, int* out_size) {
  FILE* file = fopen(filename, "rb");
  if (!file) return nullptr;
  int size = 0;
  if (fread(&size, sizeof(int), 1, file) != 1) { fclose(file); return nullptr; }
  float* buf = (float*)malloc(sizeof(float) * size);
  if (!buf) { fclose(file); return nullptr; }
  if ((int)fread(buf, sizeof(float), size, file) != size) {
    free(buf); fclose(file); return nullptr;
  }
  fclose(file);
  if (out_size) *out_size = size;
  return buf;
}
static int write_bin_with_size(const char* filename, const float* y, int N){
  FILE* f = fopen(filename, "wb"); if(!f) return -1;
  if (fwrite(&N, sizeof(int), 1, f) != 1) { fclose(f); return -1; }
  if ((int)fwrite(y, sizeof(float), N, f) != N) { fclose(f); return -1; }
  fclose(f); return 0;
}
static int compare_floats(const float* src, const float* gold, int count) {
  int num_errors = 0;
  float max_value = 0.0f;
  for (int i = 0; i < count; ++i) max_value = fmaxf(max_value, fabsf(gold[i]));
  const float abstol = 1e-4f * max_value; // same rule as OpenCL code
  for (int i = 0; i < count; ++i) {
    float diff = fabsf(gold[i] - src[i]);
    if (!(diff <= abstol || diff < 0.002f * fabsf(gold[i]))) {
      if (num_errors < 10)
        printf("Fail at row %d: (gold) %f != %f (computed)\n", i, gold[i], src[i]);
      ++num_errors;
    }
  }
  return num_errors;
}

// ---------- CUDA kernel (1:1 with OpenCL naive_kernel) ----------
__global__ void naive_kernel_cuda(float *Anext,
                                  const float *A0,
                                  int nx, int ny, int nz,
                                  float c0, float c1)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
  int j = blockIdx.y * blockDim.y + threadIdx.y + 1;  // ← Y도 일반화
  int k = blockIdx.z + 1;

  if (i < nx-1 && j < ny-1) {  // j 가드 추가
    int plane = nx * ny;
    int idx   = i + j * nx + k * plane;

    // OpenCL 과 동일: c1*이웃합 - c0*A0
    Anext[idx] =
        c1 * ( A0[idx-1] + A0[idx+1]
             + A0[idx-nx] + A0[idx+nx]
             + A0[idx-plane] + A0[idx+plane] )
      - c0 * A0[idx];
  }
}

static void usage() {
  printf("Usage: ./stencil_cuda nx ny nz t -i input.bin [-o out.bin]\n");
}

int main(int argc, char** argv) {
  if (argc < 6) { usage(); return 1; }

  int nx = atoi(argv[1]);
  int ny = atoi(argv[2]);
  int nz = atoi(argv[3]);
  int iteration = atoi(argv[4]);

  std::string in_path, out_path;
  int opt;
  // parse -i, -o
  while ((opt = getopt(argc-4, argv+4, "i:o:h")) != -1) {
    switch (opt) {
      case 'i': in_path = optarg; break;
      case 'o': out_path = optarg; break;
      case 'h': usage(); return 0;
      default : usage(); return 1;
    }
  }

  if (nx < 3 || ny < 3 || nz < 3 || iteration < 1 || in_path.empty()) {
    usage(); return 1;
  }

  float c0 = 1.0f/6.0f;
  float c1 = 1.0f/6.0f/6.0f;

  const size_t total_elems = (size_t)nx * ny * nz;
  const size_t nbytes = total_elems * sizeof(float);

  // Host buffers
  std::vector<float> h_A0(total_elems);
  std::vector<float> h_Anext(total_elems, 0.0f);

  // Read input.bin (raw floats) like the OpenCL code
  {
    FILE* fp = fopen(in_path.c_str(), "rb");
    if (!fp) { perror("fopen input"); return 1; }
    if (read_data(h_A0.data(), nx, ny, nz, fp) != 0) {
      fprintf(stderr, "Error: input size mismatch for %s\n", in_path.c_str());
      fclose(fp); return 1;
    }
    fclose(fp);
    // Anext 초기값은 A0로 복사(원본과 동일)
    memcpy(h_Anext.data(), h_A0.data(), nbytes);
  }

  // Device buffers
  float *d_A0 = nullptr, *d_Anext = nullptr;
  CUDA_CHECK(hipMalloc(&d_A0, nbytes));
  CUDA_CHECK(hipMalloc(&d_Anext, nbytes));

  // Copy initial data
  CUDA_CHECK(hipMemcpy(d_A0,   h_A0.data(),   nbytes, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_Anext,h_Anext.data(),nbytes, hipMemcpyHostToDevice));

  // Launch config mirroring OpenCL grid rounding (tx=128) and offsets (+1,+1,+1)
  const int tx = 128;
  dim3 block(tx, 1, 1);
  dim3 grid((nx - 2 + tx - 1) / tx, ny - 2, nz - 2);

  printf("grid=(%d,%d,%d), block=(%d,%d,%d)\n", grid.x, grid.y, grid.z, block.x, block.y, block.z);

  CUDA_CHECK(hipDeviceSynchronize());
  auto t0 = std::chrono::high_resolution_clock::now();

  for (int t = 0; t < iteration; ++t) {
    naive_kernel_cuda<<<grid, block>>>(d_Anext, d_A0, nx, ny, nz, c0, c1);
    CUDA_CHECK(hipGetLastError());
    // swap like OpenCL host
    float* tmp = d_A0; d_A0 = d_Anext; d_Anext = tmp;
  }

  CUDA_CHECK(hipDeviceSynchronize());
  auto t1 = std::chrono::high_resolution_clock::now();
  double ms = std::chrono::duration_cast<std::chrono::milliseconds>(t1 - t0).count();
  printf("Elapsed time: %.3f ms (for %d iterations)\n", ms, iteration);

  // In OpenCL code, they swapped one extra time after loop to make d_Anext the latest.
  // We mirror that here for parity:
  {
    float* tmp = d_A0; d_A0 = d_Anext; d_Anext = tmp;
  }

  CUDA_CHECK(hipMemcpy(h_Anext.data(), d_Anext, nbytes, hipMemcpyDeviceToHost));

  // Optional output file (Parboil style: int N + float[N])
  if (!out_path.empty()) {
    if (write_bin_with_size(out_path.c_str(), h_Anext.data(), (int)total_elems) != 0) {
      fprintf(stderr, "Failed to write %s\n", out_path.c_str());
    } else {
      printf("Wrote %s\n", out_path.c_str());
    }
  }

  // Gold compare: input.bin -> input.gold (int N; float[N])
  {
    std::string gold_path = replace_ext(in_path, ".gold");
    int gold_size = 0;
    float* gold = read_gold_file(gold_path.c_str(), &gold_size);
    if (gold) {
      if (gold_size != (int)total_elems) {
        printf("error: gold data size mismatch: current=%zu, expected=%d\n", total_elems, gold_size);
      } else {
        int errors = compare_floats(h_Anext.data(), gold, gold_size);
        if (errors > 0) printf("FAILED!\n"); else printf("PASSED!\n");
      }
      free(gold);
    } else {
      printf("(gold '%s' not found — skipping check)\n", gold_path.c_str());
    }
  }

  CUDA_CHECK(hipFree(d_A0));
  CUDA_CHECK(hipFree(d_Anext));
  return 0;
}
// nvcc -O2 -std=c++17 transpose.cu -o transpose
#include <hip/hip_runtime.h>

#include <cassert>
#include <chrono>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <string>
#include <unistd.h>
#include <vector>

#ifndef TYPE
#define TYPE float
#endif

#ifndef FLOAT_ULP
#define FLOAT_ULP 6
#endif

#define CUDA_CHECK(expr)                                                        \
  do {                                                                          \
    hipError_t _err = (expr);                                                  \
    if (_err == hipSuccess) break;                                             \
    fprintf(stderr, "CUDA Error: %s returned %d (%s) at %s:%d\n", #expr,        \
            (int)_err, hipGetErrorString(_err), __FILE__, __LINE__);           \
    std::exit(EXIT_FAILURE);                                                    \
  } while (0)

static inline bool almost_equal(float a, float b, int ulp = FLOAT_ULP) {
  union { float f; int i; } fa{a}, fb{b};
  return std::abs(fa.i - fb.i) <= ulp;
}

// CPU reference for non-square transpose
// A is width x height, row-major: A[x + y*width]
// B is height x width, row-major: B[y + x*height] = A[x + y*width]
static void transpose_cpu(TYPE* __restrict__ B,
                          const TYPE* __restrict__ A,
                          int width, int height) {
  for (int y = 0; y < height; ++y) {
    for (int x = 0; x < width; ++x) {
      B[y + x * height] = A[x + y * width];
    }
  }
}

// Kernel maps get_global_id(0/1) -> (x,y) with arbitrary block size.
// Guards handle non-square sizes.
__global__ void transpose_naive(TYPE* __restrict__ dst,
                                const TYPE* __restrict__ src,
                                int width, int height) {
  const int x = blockIdx.x * blockDim.x + threadIdx.x; // get_global_id(0)
  const int y = blockIdx.y * blockDim.y + threadIdx.y; // get_global_id(1)
  if (x < width && y < height) {
    dst[y + x * height] = src[x + y * width];
  }
}

// ------------------------------------------
// Host utils
// ------------------------------------------
static void show_usage(const char* prog) {
  printf("Usage: %s [-w width] [-H height] [-b block] [-h]\n"
         "  -w  matrix width  (default 1024)\n"
         "  -H  matrix height (default 1024)\n"
         "  -b  block size (square: b x b, default 1)\n"
         "  -h  show this help\n", prog);
}

static int gWidth  = 1024;
static int gHeight = 1024;
static int gBlock  = 1;

static void parse_args(int argc, char** argv) {
  int c;
  while ((c = getopt(argc, argv, "w:H:b:h")) != -1) {
    switch (c) {
      case 'w': gWidth  = std::atoi(optarg); break;
      case 'H': gHeight = std::atoi(optarg); break;
      case 'b': gBlock  = std::atoi(optarg); break;
      case 'h': show_usage(argv[0]); std::exit(0); break;
      default : show_usage(argv[0]); std::exit(EXIT_FAILURE);
    }
  }
  if (gWidth <= 0 || gHeight <= 0) {
    fprintf(stderr, "Error: width/height must be > 0\n");
    std::exit(EXIT_FAILURE);
  }
  if (gBlock <= 0) {
    fprintf(stderr, "Error: block must be > 0\n");
    std::exit(EXIT_FAILURE);
  }
  // Basic safety: threads per block must not exceed device limit (commonly 1024)
  if (1LL * gBlock * gBlock > 1024) {
    fprintf(stderr, "Error: block*bock=%d exceeds 1024 threads per block\n", gBlock * gBlock);
    std::exit(EXIT_FAILURE);
  }
  printf("Matrix: %d x %d, block=%d x %d\n", gWidth, gHeight, gBlock, gBlock);
}

int main(int argc, char** argv) {
  parse_args(argc, argv);
  const int width  = gWidth;
  const int height = gHeight;
  const size_t elems  = size_t(width) * size_t(height);
  const size_t nbytes = elems * sizeof(TYPE);

  // Host memory
  std::vector<TYPE> h_src(elems);
  std::vector<TYPE> h_dst(elems, TYPE(0));
  std::vector<TYPE> h_ref(elems, TYPE(0));

  // Init input
  std::srand(50);
  for (size_t i = 0; i < elems; ++i) {
    h_src[i] = static_cast<float>(std::rand()) / RAND_MAX;
  }

  // Device buffers
  TYPE *d_src = nullptr, *d_dst = nullptr;
  CUDA_CHECK(hipMalloc(&d_src, nbytes));
  CUDA_CHECK(hipMalloc(&d_dst, nbytes));

  // Upload
  CUDA_CHECK(hipMemcpy(d_src, h_src.data(), nbytes, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemset(d_dst, 0, nbytes));

  // Kernel launch params from user
  dim3 block(gBlock, gBlock, 1);
  dim3 grid( (width  + block.x - 1) / block.x,
             (height + block.y - 1) / block.y,
             1 );

  printf("grid=(%d,%d,%d), block=(%d,%d,%d)\n",
         grid.x, grid.y, grid.z, block.x, block.y, block.z);

  // Launch & time
  auto t0 = std::chrono::high_resolution_clock::now();
  transpose_naive<<<grid, block>>>(d_dst, d_src, width, height);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
  auto t1 = std::chrono::high_resolution_clock::now();
  double ms =
      std::chrono::duration_cast<std::chrono::milliseconds>(t1 - t0).count();
  printf("Elapsed time: %g ms\n", ms);

  // Download
  CUDA_CHECK(hipMemcpy(h_dst.data(), d_dst, nbytes, hipMemcpyDeviceToHost));

  // Reference & verify
  transpose_cpu(h_ref.data(), h_src.data(), width, height);
  int errors = 0;
  for (size_t i = 0; i < elems; ++i) {
    if (!almost_equal(h_dst[i], h_ref[i])) {
      if (errors < 100) {
        printf("*** error: [%zu] expected=%f, actual=%f\n",
               i, (double)h_ref[i], (double)h_dst[i]);
      }
      ++errors;
    }
  }
  if (errors == 0) {
    printf("PASSED!\n");
  } else {
    printf("FAILED! - %d errors\n", errors);
  }

  CUDA_CHECK(hipFree(d_src));
  CUDA_CHECK(hipFree(d_dst));
  return errors;
}
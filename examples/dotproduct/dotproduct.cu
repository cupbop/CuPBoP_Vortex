// dotproduct_cuda.cu
// nvcc -O3 -std=c++17 dotproduct_cuda.cu -o dotproduct_cuda

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cstdint>
#include <cstring>
#include <cassert>
#include <vector>
#include <chrono>

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 256
#endif

#define FLOAT_ULP 6

#define CUDA_CHECK(expr)                                                          \
  do {                                                                            \
    hipError_t _err = (expr);                                                    \
    if (_err != hipSuccess) {                                                    \
      fprintf(stderr, "CUDA Error: %s (%d) at %s:%d\n",                           \
              hipGetErrorString(_err), (int)_err, __FILE__, __LINE__);           \
      exit(EXIT_FAILURE);                                                         \
    }                                                                             \
  } while (0)

static bool compare_equal(float a, float b) {
  union fi_t { float f; int32_t i; };
  fi_t fa, fb;
  fa.f = a;
  fb.f = b;
  auto d = std::abs(fa.i - fb.i);
  return d <= FLOAT_ULP;
}

__global__ void dotProductKernel(const float* __restrict__ a,
                                 const float* __restrict__ b,
                                 float* __restrict__ partialSums,
                                 int n)
{
  // OpenCL의 gid, global_size와 동일한 개념을 grid-stride로 구현
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  int global_size = gridDim.x * blockDim.x;

  float sum = 0.0f;
  for (int i = gid; i < n; i += global_size) {
    sum += a[i] * b[i];
  }

  // 블록 내부 리덕션 (OpenCL의 __local + barrier와 동일)
  __shared__ float sdata[BLOCK_SIZE];
  sdata[threadIdx.x] = sum;
  __syncthreads();

  // 트리 감산
  for (int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (threadIdx.x < s) {
      sdata[threadIdx.x] += sdata[threadIdx.x + s];
    }
    __syncthreads();
  }

  // 블록 리더가 partialSums에 기록
  if (threadIdx.x == 0) {
    partialSums[blockIdx.x] = sdata[0];
  }
}

static void show_usage() {
  printf("Usage: [-n size] [-h]\n");
}

int main(int argc, char** argv) {
  // ---- parse args ----
  int n = 1 << 20; // default
  int opti = 1;
  while (opti < argc) {
    if (!strcmp(argv[opti], "-n") && opti + 1 < argc) {
      n = std::atoi(argv[++opti]);
      ++opti;
    } else if (!strcmp(argv[opti], "-h")) {
      show_usage();
      return 0;
    } else {
      show_usage();
      return 1;
    }
  }

  // ---- device info ----
  int dev = 0;
  CUDA_CHECK(hipSetDevice(dev));
  hipDeviceProp_t prop{};
  CUDA_CHECK(hipGetDeviceProperties(&prop, dev));
  printf("Using device: %s\n", prop.name);
  printf("Vector size n=%d\n", n);

  // ---- host buffers ----
  std::vector<float> h_a(n), h_b(n);

  // 입력 데이터 생성 (OpenCL 버전과 동일한 랜덤 생성)
  std::srand(0xC0FFEE);
  for (int i = 0; i < n; ++i) {
    h_a[i] = static_cast<float>(std::rand()) / RAND_MAX;
    h_b[i] = static_cast<float>(std::rand()) / RAND_MAX;
  }

  // ---- device buffers ----
  float *d_a = nullptr, *d_b = nullptr, *d_partial = nullptr;

  // 그리드/블록 구성 (OpenCL의 글로벌/로컬 크기 대응)
  int block = BLOCK_SIZE;
  // OpenCL처럼 "글로벌 사이즈 = 블록*그리드"로 잡되,
  // 너무 많은 블록을 만들지 않도록 디바이스 멀티프로세서 수를 고려
  //int maxBlocks = prop.multiProcessorCount * 32; // 적당한 상한
  int grid = (n + block - 1) / block;
  //if (grid > maxBlocks) grid = maxBlocks;
  if (grid < 1) grid = 1;

  size_t bytes = sizeof(float) * n;
  size_t partial_bytes = sizeof(float) * grid;

  CUDA_CHECK(hipMalloc(&d_a, bytes));
  CUDA_CHECK(hipMalloc(&d_b, bytes));
  CUDA_CHECK(hipMalloc(&d_partial, partial_bytes));

  // ---- upload ----
  CUDA_CHECK(hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice));

  // ---- run kernel ----
  printf("Launch kernel: grid=%d block=%d\n", grid, block);
  auto t0 = std::chrono::high_resolution_clock::now();
  dotProductKernel<<<grid, block>>>(d_a, d_b, d_partial, n);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
  auto t1 = std::chrono::high_resolution_clock::now();
  double ms = std::chrono::duration_cast<std::chrono::milliseconds>(t1 - t0).count();
  printf("Elapsed time: %g ms\n", ms);

  // ---- read back partial sums ----
  std::vector<float> h_partial(grid);
  CUDA_CHECK(hipMemcpy(h_partial.data(), d_partial, partial_bytes, hipMemcpyDeviceToHost));

  // ---- final reduce on host (OpenCL와 동일한 2-pass) ----
  float gpu_result = 0.0f;
  for (int i = 0; i < grid; ++i) gpu_result += h_partial[i];

  // ---- CPU reference ----
  double cpu_result = 0.0;
  for (int i = 0; i < n; ++i) cpu_result += static_cast<double>(h_a[i]) * static_cast<double>(h_b[i]);

  printf("GPU result: %.9g\n", gpu_result);
  printf("CPU result: %.9g\n", cpu_result);

  // ---- check ----
  bool ok = compare_equal(gpu_result, static_cast<float>(cpu_result));
  if (!ok) {
    // 허용 오차 조금 넉넉히 보기
    float diff = std::abs(gpu_result - static_cast<float>(cpu_result));
    float rel  = diff / (std::abs(static_cast<float>(cpu_result)) + 1e-7f);
    ok = (rel < 1e-5f);
  }
  if (ok) {
    printf("PASSED!\n");
  } else {
    printf("FAILED!\n");
  }

  // ---- cleanup ----
  CUDA_CHECK(hipFree(d_a));
  CUDA_CHECK(hipFree(d_b));
  CUDA_CHECK(hipFree(d_partial));
  return ok ? 0 : 1;
}